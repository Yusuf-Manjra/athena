/*
// Copyright (C) 2002-2022 CERN for the benefit of the ATLAS collaboration
*/

#include "CaloRecGPU/DataHolders.h"

void ConstantDataHolder::sendToGPU(const bool clear_CPU)
{
  m_cell_noise_dev = m_cell_noise;
  m_geometry_dev = m_geometry;
  if (clear_CPU)
    {
      m_cell_noise.clear();
      m_geometry.clear();
    }
}

void EventDataHolder::sendToGPU(const bool clear_CPU, const bool has_state, const bool has_clusters, const bool has_pairs)
{
  m_cell_info_dev = m_cell_info;
  if (has_state)
    {
      m_cell_state_dev = m_cell_state;
    }
  else
    {
      m_cell_state_dev.allocate();
    }
  if (has_clusters)
    {
      m_clusters_dev = m_clusters;
    }
  else
    {
      m_clusters_dev.allocate();
    }
  if (has_pairs)
    {
      m_pairs_dev = m_pairs;
    }
  else
    {
      m_pairs_dev.allocate();
    }

  if (!has_clusters)
    {
      hipMemset(&(m_clusters_dev->number), 0, sizeof(m_clusters_dev->number));
    }
  if (!has_pairs)
    {
      hipMemset(&(m_pairs_dev->number), 0, sizeof(m_pairs_dev->number));
      hipMemset(&(m_pairs_dev->reverse_number), 0, sizeof(m_pairs_dev->reverse_number));
    }
  //We're not doing this through hipMemsetAsync because it is reasonable to expect
  //the clusters to be fully sent before doing any more operations.

  if (clear_CPU)
    {
      m_cell_info.clear();
      m_cell_state.clear();
      m_pairs.clear();
    }
}

void EventDataHolder::returnToCPU(const bool clear_GPU, const bool return_clusters)
{
  m_cell_state = m_cell_state_dev;
  if (return_clusters)
    {
      m_clusters = m_clusters_dev;
    }
  if (clear_GPU)
    {
      m_cell_state_dev.clear();
      m_clusters_dev.clear();
      m_pairs_dev.clear();
      m_cell_info_dev.clear();
    }
}

void EventDataHolder::allocate(const bool also_GPU)
{
  m_cell_info.allocate();
  m_cell_state.allocate();
  m_pairs.allocate();
  m_clusters.allocate();

  if (also_GPU)
    {
      m_cell_info_dev.allocate();
      m_cell_state_dev.allocate();
      m_pairs_dev.allocate();
      m_clusters_dev.allocate();
    }
}