// Dear emacs, this is -*- c++ -*-
/*
// Copyright (C) 2002-2021 CERN for the benefit of the ATLAS collaboration
*/

#include "CaloRecGPU/Helpers.h"

void * CaloRecGPU::CUDA_Helpers::allocate(const size_t num)
{
  void * ret;
  CUDA_ERRCHECK(hipMalloc(&ret, num));
  return ret;
}

void CaloRecGPU::CUDA_Helpers::deallocate(void * address)
{
  CUDA_ERRCHECK(hipFree(address));
}

void CaloRecGPU::CUDA_Helpers::GPU_to_CPU(void * dest, const void * const source, const size_t num)
{
  CUDA_ERRCHECK(hipMemcpy(dest, source, num, hipMemcpyDeviceToHost));
}

void CaloRecGPU::CUDA_Helpers::CPU_to_GPU(void * dest, const void * const source, const size_t num)
{
  CUDA_ERRCHECK(hipMemcpy(dest, source, num, hipMemcpyHostToDevice));
}

void CaloRecGPU::CUDA_Helpers::GPU_to_GPU(void * dest, const void * const source, const size_t num)
{
  CUDA_ERRCHECK(hipMemcpy(dest, source, num, hipMemcpyDeviceToDevice));
}