#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "BasicGPUClusterInfoCalculatorImpl.h"


#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>

using namespace CaloRecGPU;

constexpr static int SeedCellPropertiesBlockSize = 512;

constexpr static int CalculateClusterInfoBlockSize = 320;
constexpr static int FinalizeClusterInfoBlockSize = 256;
constexpr static int ClearInvalidCellsBlockSize = 512;

#if defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ > 350
  #if CUDART_VERSION >= 12000
    #define CAN_USE_TAIL_LAUNCH 1
  #else
    #define CAN_USE_TAIL_LAUNCH 0
  #endif
#elif defined(__CUDA_ARCH__)
  #error "CUDA compute capability at least 3.5 is needed so we can have dynamic parallelism!"
#endif

/**********************************************************************************/

__global__ static
void seedCellPropertiesKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                               Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries,
                               const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                               const int cluster_number)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < cluster_number)
    {
      clusters_arr->clusterEnergy[i] = 0.f;
      clusters_arr->clusterEt[i] = 0.f;
      clusters_arr->clusterEta[i] = 0.f;
      clusters_arr->clusterPhi[i] = 0.f;
      const int seed_cell = clusters_arr->seedCellID[i];
      if (seed_cell >= 0)
        {
          temporaries->seedCellPhi[i] = geometry->phi[seed_cell];
        }
      else
        {
          temporaries->seedCellPhi[i] = 0.f;
        }
    }
}


__global__ static
void seedCellPropertiesDeferKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                    Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries,
                                    const Helpers::CUDA_kernel_object<GeometryArr> geometry)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    {
      const int cluster_number = clusters_arr->number;

      const int i_dimBlock = SeedCellPropertiesBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(cluster_number, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);
#if CAN_USE_TAIL_LAUNCH
      seedCellPropertiesKernel <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(clusters_arr, temporaries, geometry, cluster_number);
#else
      seedCellPropertiesKernel <<< dimGrid, dimBlock>>>(clusters_arr, temporaries, geometry, cluster_number);
#endif
    }
}

void updateSeedCellProperties(CaloRecGPU::EventDataHolder & holder,
                              CaloRecGPU::Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temps,
                              const ConstantDataHolder & instance_data, const bool synchronize,
                              CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  seedCellPropertiesDeferKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_clusters_dev, temps, instance_data.m_geometry_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/**********************************************************************************/

__device__  static inline
float regularize_angle(const float b, const float a)
//a. k. a. proxim in Athena code.
{
  const float diff = b - a;
  const float divi = (fabsf(diff) - Helpers::Constants::pi<float>) / (2 * Helpers::Constants::pi<float>);
  return b - ceilf(divi) * ((b > a + Helpers::Constants::pi<float>) - (b < a - Helpers::Constants::pi<float>)) * 2 * Helpers::Constants::pi<float>;
}

__global__ static
void calculateClusterInfoKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                 const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                 const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                 const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                 const Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const ClusterTag tag = cell_state_arr->clusterTag[index];
      if (tag.is_part_of_cluster())
        //By this point they all have the terminals anyway, so...
        {
          if (tag.is_shared_between_clusters())
            {
              const int primary_cluster = tag.cluster_index();
              const int secondary_cluster = tag.secondary_cluster_index();

              const float secondary_weight = __int_as_float(tag.secondary_cluster_weight());
              const float weight = 1.0f - secondary_weight;

              const float energy = cell_info_arr->energy[index];
              const float abs_energy = fabsf(energy);
              const float phi_raw = geometry->phi[index];

              atomicAdd(&(clusters_arr->clusterEnergy[primary_cluster]), energy * weight);
              atomicAdd(&(clusters_arr->clusterEt[primary_cluster]), abs_energy * weight);
              atomicAdd(&(clusters_arr->clusterEta[primary_cluster]), abs_energy * geometry->eta[index] * weight);

              const float primary_phi_0 = temporaries->seedCellPhi[primary_cluster];
              const float primary_phi_real = regularize_angle(phi_raw, primary_phi_0);
              atomicAdd(&(clusters_arr->clusterPhi[primary_cluster]), primary_phi_real * abs_energy * weight);

              atomicAdd(&(clusters_arr->clusterEnergy[secondary_cluster]), energy * secondary_weight);
              atomicAdd(&(clusters_arr->clusterEt[secondary_cluster]), abs_energy * secondary_weight);
              atomicAdd(&(clusters_arr->clusterEta[secondary_cluster]), abs_energy * geometry->eta[index] * secondary_weight);

              const float secondary_phi_0 = temporaries->seedCellPhi[secondary_cluster];
              const float secondary_phi_real = regularize_angle(phi_raw, secondary_phi_0);
              atomicAdd(&(clusters_arr->clusterPhi[secondary_cluster]), secondary_phi_real * abs_energy * secondary_weight);
            }
          else
            {
              const int cluster_index = tag.cluster_index();
              const float energy = cell_info_arr->energy[index];
              const float abs_energy = fabsf(energy);
              const float phi_raw = geometry->phi[index];

              atomicAdd(&(clusters_arr->clusterEnergy[cluster_index]), energy);
              atomicAdd(&(clusters_arr->clusterEt[cluster_index]), abs_energy);
              atomicAdd(&(clusters_arr->clusterEta[cluster_index]), abs_energy * geometry->eta[index]);

              const float phi_0 = temporaries->seedCellPhi[cluster_index];
              const float phi_real = regularize_angle(phi_raw, phi_0);
              atomicAdd(&(clusters_arr->clusterPhi[cluster_index]), phi_real * abs_energy);
            }
        }
    }
}


__global__ static
void finalizeClusterInfoKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int cluster_number,
                                const bool cut_in_absolute_ET, const float ET_threshold                             )
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < cluster_number)
    {

      const float abs_energy = clusters_arr->clusterEt[i];

      if (abs_energy > 0)
        {
          const float tempeta = clusters_arr->clusterEta[i] / abs_energy;

          clusters_arr->clusterEta[i] = tempeta;

          const float temp_ET = clusters_arr->clusterEnergy[i] / coshf(abs(tempeta));

          clusters_arr->clusterEt[i] = temp_ET;

          clusters_arr->clusterPhi[i] = regularize_angle(clusters_arr->clusterPhi[i] / abs_energy, 0.f);

          if ( !(temp_ET > ET_threshold || (cut_in_absolute_ET && fabsf(temp_ET) > ET_threshold) ) )
            {
              clusters_arr->seedCellID[i] = -1;
            }
        }
      else
        {
          clusters_arr->seedCellID[i] = -1;
          //This is just a way to signal that this is an invalid cluster.
        }
    }
}

__global__ static
void finalizeClustersDeferKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                  const bool cut_in_absolute_ET, const float ET_threshold)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    {
      const int cluster_number = clusters_arr->number;

      const int i_dimBlock = FinalizeClusterInfoBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(cluster_number, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);
#if CAN_USE_TAIL_LAUNCH
      finalizeClusterInfoKernel <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(clusters_arr, cluster_number, cut_in_absolute_ET, ET_threshold);
#else
      finalizeClusterInfoKernel <<< dimGrid, dimBlock>>>(clusters_arr, cluster_number, cut_in_absolute_ET, ET_threshold);
#endif
    }
}


__global__ static
void clearInvalidCells(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                       const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const ClusterTag tag = cell_state_arr->clusterTag[index];
      if (tag.is_part_of_cluster())
        //By this point they all have the terminals anyway, so...
        {
          if (tag.is_shared_between_clusters())
            {
              const int first_cluster = tag.cluster_index();
              const int second_cluster = tag.secondary_cluster_index();

              const int first_seed = clusters_arr->seedCellID[first_cluster];
              const int second_seed = clusters_arr->seedCellID[second_cluster];

              if (first_seed < 0 && second_seed < 0)
                {
                  cell_state_arr->clusterTag[index] = ClusterTag:: make_invalid_tag();
                }
              else if (first_seed < 0)
                {
                  cell_state_arr->clusterTag[index] = ClusterTag::make_tag(second_cluster);
                }
              else if (second_seed < 0)
                {
                  cell_state_arr->clusterTag[index] = ClusterTag::make_tag(first_cluster);
                }
              else /*if (first_seed >= 0 && second_seed >= 0)*/
                {
                  //Do nothing: the tag's already OK.
                }
            }
          else
            {
              if (clusters_arr->seedCellID[tag.cluster_index()] < 0)
                {
                  cell_state_arr->clusterTag[index] = ClusterTag:: make_invalid_tag();
                }
            }
        }
    }
}

void calculateClusterProperties(CaloRecGPU::EventDataHolder & holder,
                                CaloRecGPU::Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temps,
                                const ConstantDataHolder & instance_data, const bool synchronize,
                                const bool cut_in_absolute_ET, const float ET_threshold,
                                CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  const int i_dimBlock1 = CalculateClusterInfoBlockSize;
  const int i_dimGrid1 = Helpers::int_ceil_div(NCaloCells, i_dimBlock1);
  const dim3 dimBlock1(i_dimBlock1, 1, 1);
  const dim3 dimGrid1(i_dimGrid1, 1, 1);

  const int i_dimBlock2 = ClearInvalidCellsBlockSize;
  const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells, i_dimBlock2);
  const dim3 dimBlock2(i_dimBlock2, 1, 1);
  const dim3 dimGrid2(i_dimGrid2, 1, 1);

  calculateClusterInfoKernel <<< dimGrid1, dimBlock1, 0, stream_to_use>>>(holder.m_clusters_dev, holder.m_cell_state_dev,
                                                                          holder.m_cell_info_dev, instance_data.m_geometry_dev, temps);

  finalizeClustersDeferKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_clusters_dev, cut_in_absolute_ET, ET_threshold);

  clearInvalidCells <<< dimGrid2, dimBlock2, 0, stream_to_use>>>(holder.m_cell_state_dev, holder.m_clusters_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}