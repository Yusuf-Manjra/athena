#include "hip/hip_runtime.h"
// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2022 CERN for the benefit of the ATLAS collaboration
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "BasicGPUClusterInfoCalculatorImpl.h"


#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>


using namespace CaloRecGPU;

void BasicGPUClusterInfoCalculatorTemporariesHolder::allocate()
{
  m_temporaries_dev.allocate();
}

constexpr static int SeedCellPropertiesBlockSize = 512;

constexpr static int CalculateClusterInfoBlockSize = 320;
constexpr static int FinalizeClusterInfoBlockSize = 256;

/**********************************************************************************/

__global__ static
void seedCellPropertiesKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                               Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries,
                               const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                               const int cluster_number)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < cluster_number)
    {
      clusters_arr->clusterEnergy[i] = 0.f;
      clusters_arr->clusterEt[i] = 0.f;
      clusters_arr->clusterEta[i] = 0.f;
      clusters_arr->clusterPhi[i] = 0.f;
      temporaries->seedCellPhi[i] = geometry->phi[clusters_arr->seedCellID[i]];
    }
}


__global__ static
void seedCellPropertiesDeferKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                    Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries,
                                    const Helpers::CUDA_kernel_object<GeometryArr> geometry)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    {
      const int cluster_number = clusters_arr->number;

      const int i_dimBlock = SeedCellPropertiesBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(cluster_number, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);

      seedCellPropertiesKernel <<< dimGrid, dimBlock >>>(clusters_arr, temporaries, geometry, cluster_number);

    }
}

void updateSeedCellProperties(EventDataHolder & holder, BasicGPUClusterInfoCalculatorTemporariesHolder & temps,
                              const ConstantDataHolder & instance_data, const bool synchronize)
{
  seedCellPropertiesDeferKernel <<<1, 1>>>(holder.m_clusters_dev, temps.m_temporaries_dev, instance_data.m_geometry_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}


/**********************************************************************************/

__device__  static inline
float regularize_angle(const float b, const float a)
//a. k. a. proxim in Athena code.
{
  const float diff = b - a;
  const float divi = (fabsf(diff) - Helpers::Constants::pi<float>) / (2 * Helpers::Constants::pi<float>);
  return b - ceilf(divi) * ((b > a + Helpers::Constants::pi<float>) - (b < a - Helpers::Constants::pi<float>)) * 2 * Helpers::Constants::pi<float>;
}

__global__ static
void calculateClusterInfoKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                 const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                 const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                 const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                 const Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const tag_type tag = cell_state_arr->clusterTag[index];
      if (Tags::is_part_of_cluster(tag))
        //By this point they all have the terminals anyway, so...
        {
          const int cluster_index = Tags::get_index_from_tag(tag);
          const float energy = cell_info_arr->energy[index];
          const float abs_energy = fabsf(energy);
          const float phi_raw = geometry->phi[index];

          atomicAdd(&(clusters_arr->clusterEnergy[cluster_index]), energy);
          atomicAdd(&(clusters_arr->clusterEt[cluster_index]), abs_energy);
          atomicAdd(&(clusters_arr->clusterEta[cluster_index]), abs_energy * geometry->eta[index]);

          const float phi_0 = temporaries->seedCellPhi[cluster_index];
          const float phi_real = regularize_angle(phi_raw, phi_0);
          atomicAdd(&(clusters_arr->clusterPhi[cluster_index]), phi_real * abs_energy);

        }
    }
}


__global__ static
void finalizeClusterInfoKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int cluster_number)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < cluster_number)
    {

      const float abs_energy = clusters_arr->clusterEt[i];

      if (abs_energy > 0)
        {
          const float tempeta = clusters_arr->clusterEta[i] / abs_energy;

          clusters_arr->clusterEta[i] = tempeta;

          clusters_arr->clusterEt[i] = clusters_arr->clusterEnergy[i] / coshf(abs(tempeta));

          clusters_arr->clusterPhi[i] = regularize_angle(clusters_arr->clusterPhi[i] / abs_energy, 0.f);
        }
      else
        {
          clusters_arr->seedCellID[i] = -1;
          //This is just a way to signal that this is an invalid cluster.
        }
    }
}

__global__ static
void finalizeClustersDeferKernel( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    {
      const int cluster_number = clusters_arr->number;

      const int i_dimBlock = FinalizeClusterInfoBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(cluster_number, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);

      finalizeClusterInfoKernel <<< dimGrid, dimBlock >>>(clusters_arr, cluster_number);

    }
}

void calculateClusterProperties(EventDataHolder & holder, BasicGPUClusterInfoCalculatorTemporariesHolder & temps,
                                const ConstantDataHolder & instance_data, const bool synchronize)
{

  const int i_dimBlock = CalculateClusterInfoBlockSize;
  const int i_dimGrid = Helpers::int_ceil_div(NCaloCells, i_dimBlock);
  const dim3 dimBlock(i_dimBlock, 1, 1);
  const dim3 dimGrid(i_dimGrid, 1, 1);

  calculateClusterInfoKernel <<< dimGrid, dimBlock>>>(holder.m_clusters_dev, holder.m_cell_state_dev,
                                                      holder.m_cell_info_dev, instance_data.m_geometry_dev, temps.m_temporaries_dev);

  finalizeClustersDeferKernel <<< 1, 1>>>(holder.m_clusters_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}