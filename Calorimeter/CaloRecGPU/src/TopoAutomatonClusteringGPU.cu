#include "hip/hip_runtime.h"
// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2022 CERN for the benefit of the ATLAS collaboration
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "TopoAutomatonClusteringGPU.h"


#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>

using namespace CaloRecGPU;

void TACTemporariesHolder::allocate()
{
  m_temporaries_dev.allocate();
}

void TACOptionsHolder::allocate()
{
  m_options.allocate();
  m_options_dev.allocate();
}

void TACOptionsHolder::sendToGPU(const bool clear_CPU)
{
  m_options_dev = m_options;
  if (clear_CPU)
    {
      m_options.clear();
    }
}

//constexpr static int DefaultBlockSize = 256;

constexpr static int SignalToNoiseBlockSize = 512;
constexpr static int CellPairsBlockSize = 64;
constexpr static int ClusterGrowingPropagationBlockSize = 64;
constexpr static int ClusterGrowingMergingBlockSize = 512;

/******************************************************************************
 * Kernel to compute the cells signal (Energy) to noise ratio, used for the
 * cells clustering step, to define the seed cells, the growing cells and the
 * border cells.
 * It also apply the energy thresholds (seed-4, grow-2, border-0)
 ******************************************************************************/


__device__ inline static tag_type calculateTag(const float SNR, const int seed_cell_index, const int address)
{
  return Tags::make_seed_tag(__float_as_int(SNR), seed_cell_index, address);
}


static __global__
void signalToNoiseKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                          Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                          Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                          const Helpers::CUDA_kernel_object<CellNoiseArr> noise_arr,
                          const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                          const Helpers::CUDA_kernel_object<TopoAutomatonOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;


  if (index < NCaloCells)
    {
      const float cellEnergy = cell_info_arr->energy[index];
      const int gain = cell_info_arr->gain[index];

      if (GainConversion::is_invalid_cell(gain))
        {
          cell_state_arr->clusterTag[index] = Tags::InvalidTag;
          temporaries->secondaryArray[index] = Tags::InvalidTag;
          return;
        }


      float sigNoiseRatio = 0.00001f;
      //It's what's done in the CPU implementation...
      if (GainConversion::is_normal_cell(gain) || GainConversion::is_invalid_seed_cell(gain))
        {
          const int corr_gain = GainConversion::recover_invalid_seed_cell_gain(gain);
          const float cellNoise = noise_arr->noise[corr_gain][index];
          if (isfinite(cellNoise) && cellNoise > 0.0f)
            {
              sigNoiseRatio = cellEnergy / cellNoise;
            }
        }

      const float absRatio = fabsf(sigNoiseRatio);

      if (sigNoiseRatio > opts->seed_threshold || (opts->abs_seed && absRatio > opts->seed_threshold)) // is Seed
        {
          if (GainConversion::is_invalid_seed_cell(gain))
            //If the cell is just ineligible to be a seed, it can still be a growing cell.
            {
              cell_state_arr->clusterTag[index] = Tags::GrowTag;
              temporaries->secondaryArray[index] = Tags::GrowTag;
            }
          else if ( !opts->uses_sampling(geometry->caloSample[index]) )
            //If the seed belongs to a sampling that's invalid for seeds...
            {
              cell_state_arr->clusterTag[index] = Tags::GrowTag;
              temporaries->secondaryArray[index] = Tags::GrowTag;
            }
          else
            {
              const int n = atomicAdd(&(clusters_arr->number), 1);

              const tag_type tag = calculateTag(absRatio, index, n);
              //Since seed_threshold will be positive,
              //no problem with using abs here always:
              //when actually using the absolute value,
              //it's what we want, when not, cells with
              //negative SNR will not be acceptable seeds.
              cell_state_arr->clusterTag[index] = tag;
              temporaries->secondaryArray[index] = tag;

              clusters_arr->seedCellID[n] = index;
              /*clusters_arr->clusterEnergy[n] = 0.f;
              clusters_arr->clusterEt[n] = 0.f;
              clusters_arr->clusterEta[n] = 0.f;
              clusters_arr->clusterPhi[n] = 0.f;*/
              //This will be zeroed out later by the property calculation tool.
              temporaries->mergeTable[n] = Tags::clear_counter(tag);
            }
        }
      else if (sigNoiseRatio > opts->grow_threshold || (opts->abs_grow && absRatio > opts->grow_threshold)) //is Grow
        {
          cell_state_arr->clusterTag[index] = Tags::GrowTag;
          temporaries->secondaryArray[index] = Tags::GrowTag;
        }
      else if (sigNoiseRatio > opts->terminal_threshold || (opts->abs_terminal && absRatio > opts->terminal_threshold)) //is Terminal
        {
          cell_state_arr->clusterTag[index] = Tags::TerminalTag;
          temporaries->secondaryArray[index] = Tags::TerminalTag;
        }
      else //is invalid for propagation
        {
          cell_state_arr->clusterTag[index] = Tags::InvalidTag;
          temporaries->secondaryArray[index] = Tags::InvalidTag;
        }
    }
}

//run the kernel
void signalToNoise(EventDataHolder & holder, TACTemporariesHolder & temps,
                   const ConstantDataHolder & instance_data, const TACOptionsHolder & options, const bool synchronize)
{
  
  hipMemsetAsync(&(holder.m_clusters_dev->number), 0, sizeof(holder.m_clusters_dev->number), hipStreamPerThread);
  
  const int i_dimBlock = SignalToNoiseBlockSize;
  const int i_dimGrid = Helpers::int_ceil_div(NCaloCells, i_dimBlock);

  const dim3 dimBlock(i_dimBlock, 1, 1);
  const dim3 dimGrid(i_dimGrid, 1, 1);
  signalToNoiseKernel <<< dimGrid, dimBlock>>>(holder.m_cell_state_dev, holder.m_clusters_dev, temps.m_temporaries_dev,
                                               holder.m_cell_info_dev, instance_data.m_cell_noise_dev, instance_data.m_geometry_dev,
                                               options.m_options_dev);
  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}


/******************************************************************************
 * Kernel to generate the cell pairs for the growing algorithm.
 ******************************************************************************/


static __global__
void cellPairsKernel( Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                      const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                      const Helpers::CUDA_kernel_object<GeometryArr> geometry)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;


  if (index < NCaloCells)
    {

      const tag_type this_tag = cell_state_arr->clusterTag[index];

      if (Tags::is_growing_or_seed(this_tag))
        {
          const int num_neighs = geometry->nNeighbours[index];

          int neighbourList[NMaxNeighbours];
          int num_bad_neighs = 0;
          for (int i = 0; i < num_neighs; ++i)
            {
              const int neigh_ID = geometry->neighbours[index][i];
              const tag_type neigh_tag = cell_state_arr->clusterTag[neigh_ID];
              if (Tags::is_valid(neigh_tag))
                {
                  neighbourList[i - num_bad_neighs] = neigh_ID;
                }
              else
                {
                  ++num_bad_neighs;
                }
            }

          const int real_neighs = num_neighs - num_bad_neighs;
          const int n = atomicAdd(&(neighbour_pairs->number), real_neighs);
          for (int i = 0; i < real_neighs; ++i)
            {
              neighbour_pairs->cellID[n + i] = index;
              neighbour_pairs->neighbourID[n + i] = neighbourList[i];
            }
        }
      else if (Tags::is_terminal(this_tag))
        {
          const int num_neighs = geometry->nNeighbours[index];

          int neighbourList[NMaxNeighbours];
          int num_bad_neighs = 0;
          for (int i = 0; i < num_neighs; ++i)
            {
              const int neigh_ID = geometry->neighbours[index][i];
              const tag_type neigh_tag = cell_state_arr->clusterTag[neigh_ID];
              if (Tags::is_growing_or_seed(neigh_tag))
                {
                  neighbourList[i - num_bad_neighs] = neigh_ID;
                }
              else
                {
                  ++num_bad_neighs;
                }
            }

          const int real_neighs = num_neighs - num_bad_neighs;
          const int n = atomicAdd(&(neighbour_pairs->reverse_number), real_neighs);
          const int real_start = NMaxPairs - n - real_neighs;
          for (int i = 0; i < real_neighs; ++i)
            {
              neighbour_pairs->cellID[real_start + i] = index;
              neighbour_pairs->neighbourID[real_start + i] = neighbourList[i];
            }
        }
    }
}

//run the kernel
void cellPairs(EventDataHolder & holder, TACTemporariesHolder & temps,
               const ConstantDataHolder & instance_data, const TACOptionsHolder & options, const bool synchronize)
{
  hipMemsetAsync(&(holder.m_pairs_dev->number), 0, sizeof(holder.m_pairs_dev->number), hipStreamPerThread);
  hipMemsetAsync(&(holder.m_pairs_dev->reverse_number), 0, sizeof(holder.m_pairs_dev->reverse_number), hipStreamPerThread);

  const int i_dimBlock = CellPairsBlockSize;
  const int i_dimGrid = Helpers::int_ceil_div(NCaloCells, i_dimBlock);

  const dim3 dimBlock(i_dimBlock, 1, 1);
  const dim3 dimGrid(i_dimGrid, 1, 1);

  cellPairsKernel <<< dimGrid, dimBlock>>>(holder.m_pairs_dev, holder.m_cell_state_dev, instance_data.m_geometry_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}



/******************************************************************************
 * Series of kernels for the growing algorithm!
 ******************************************************************************/

__global__ static
void propagateNeighbours( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                          const int pair_number,
                          const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < pair_number)
    {
      const int this_ID = neighbour_pairs->cellID[index];
      const int neigh_ID = neighbour_pairs->neighbourID[index];

      const tag_type neigh_raw_tag = cell_state_arr->clusterTag[neigh_ID];

      const tag_type neigh_prop_tag = Tags::set_for_propagation(neigh_raw_tag);

      const tag_type this_old_raw_tag = atomicMax(&(temporaries->secondaryArray[this_ID]), neigh_prop_tag);
      if (Tags::is_part_of_cluster(this_old_raw_tag) && Tags::is_part_of_cluster(neigh_raw_tag))
        {
          //If the cell was already part of a cluster,
          //we must merge the two of them.
          //Else, we keep growing.
          const int this_address = Tags::get_index_from_tag(this_old_raw_tag);
          const int neigh_address = Tags::get_index_from_tag(neigh_raw_tag);
          if (this_address != neigh_address)
            {
              const tag_type maximum_cluster = max(Tags::clear_counter(this_old_raw_tag), Tags::clear_counter(neigh_raw_tag));
              atomicMax(&(temporaries->mergeTable[this_address]), maximum_cluster);
              atomicMax(&(temporaries->mergeTable[neigh_address]), maximum_cluster);
              temporaries->continueFlag = 1;
            }
        }
      else if (Tags::is_part_of_cluster(neigh_raw_tag))
        {
          temporaries->continueFlag = 1;
        }
    }
}

__global__ static
void mergeClusters( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                    Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const tag_type old_tag = temporaries->secondaryArray[index];
      if (Tags::is_part_of_cluster(old_tag))
        {
          const int address = Tags::get_index_from_tag(old_tag);
          const tag_type new_tag = temporaries->mergeTable[address];
          const tag_type final_tag = Tags::update_non_terminal_tag(old_tag, new_tag);
          cell_state_arr->clusterTag[index] = final_tag;
          temporaries->secondaryArray[index] = final_tag;
        }
    }
}

__global__ static
void propagateTerminals( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                         const int reverse_pair_number,
                         const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < reverse_pair_number)
    {
      const int start_index = NMaxPairs - reverse_pair_number;
      const int this_ID = neighbour_pairs->cellID[start_index + index];
      const int neigh_ID = neighbour_pairs->neighbourID[start_index + index];

      const tag_type neigh_raw_tag = cell_state_arr->clusterTag[neigh_ID];

      atomicMax(&(temporaries->secondaryArray[this_ID]), Tags::set_for_terminal_propagation(neigh_raw_tag));
    }
}

__global__ static
void finalizeClusterAttribution(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                const Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const tag_type old_tag = temporaries->secondaryArray[index];
      cell_state_arr->clusterTag[index] = Tags::clear_counter(old_tag);
    }
}

__global__ static
void clusterGrowingKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                           Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                           const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int pairs_number = neighbour_pairs->number;
      const int reverse_pairs_number = neighbour_pairs->reverse_number;

      const int i_dimBlock1 = ClusterGrowingPropagationBlockSize;
      const int i_dimGrid1 = Helpers::int_ceil_div(pairs_number, i_dimBlock1);
      const dim3 dimBlock1(i_dimBlock1, 1, 1);
      const dim3 dimGrid1(i_dimGrid1, 1, 1);

      const int i_dimBlock2 = ClusterGrowingMergingBlockSize;
      const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells, i_dimBlock2);
      const dim3 dimBlock2(i_dimBlock2, 1, 1);
      const dim3 dimGrid2(i_dimGrid2, 1, 1);
      
      const int i_dimBlock3 = ClusterGrowingPropagationBlockSize;
      const int i_dimGrid3 = Helpers::int_ceil_div(reverse_pairs_number, i_dimBlock3);
      const dim3 dimBlock3(i_dimBlock3, 1, 1);
      const dim3 dimGrid3(i_dimGrid3, 1, 1);

      temporaries->continueFlag = 1;

      while (temporaries->continueFlag > 0)
        {
          temporaries->continueFlag = 0;
          propagateNeighbours <<< dimGrid1, dimBlock1>>>(cell_state_arr, temporaries, pairs_number, neighbour_pairs);
          mergeClusters <<< dimGrid2, dimBlock2>>>(cell_state_arr, temporaries);
          if (temporaries->continueFlag == 0)
            {
              hipDeviceSynchronize();
            }
        }
      propagateTerminals <<< dimGrid3, dimBlock3>>>(cell_state_arr, temporaries, reverse_pairs_number, neighbour_pairs);
      finalizeClusterAttribution <<< dimGrid2, dimBlock2>>>(cell_state_arr, temporaries);
    }
}

//run the kernel
void clusterGrowing(EventDataHolder & holder, TACTemporariesHolder & temps,
                    const ConstantDataHolder & instance_data, const TACOptionsHolder & options, const bool synchronize)
{

  clusterGrowingKernel <<< 1, 1>>>(holder.m_cell_state_dev, temps.m_temporaries_dev, holder.m_pairs_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}
