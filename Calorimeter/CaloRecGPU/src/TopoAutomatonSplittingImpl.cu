#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "TopoAutomatonSplittingImpl.h"

#include "CaloIdentifier/LArNeighbours.h"
//It's just a struct.


#include "CLHEP/Units/SystemOfUnits.h"
//Probably will also work, given that it's just constexpr stuff.

#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>


using namespace CaloRecGPU;

void TASOptionsHolder::allocate()
{
  m_options.allocate();
  m_options_dev.allocate();
}

void TASOptionsHolder::sendToGPU(const bool clear_CPU)
{
  m_options_dev = m_options;
  if (clear_CPU)
    {
      m_options.clear();
    }
}

constexpr static int FillNeighboursFirstBlockSize = 128;
constexpr static int FillNeighboursSecondBlockSize = 128;

constexpr static int CountInferiorNeighsBlockSize = 256;
constexpr static int FindLocalMaximaBlockSize = 512;
constexpr static int FixClustersWithoutMaximaBlockSize = 512;

constexpr static int PrepareArrayForSecondaryMaximaBlockSize = 512;
constexpr static int ExcludeMaximaPropagationBlockSize = 256;
constexpr static int ResetAndCleanSecondariesBlockSize = 512;

constexpr static int PropagateSplitTagsBlockSize = 256;
constexpr static int HandleSplitIndexChangesBlockSize = 256;
constexpr static int HandleSplitTagChangesBlockSize = 256;

constexpr static int SumCellsBlockSize = 320;
constexpr static int CalculateCentroidsBlockSize = 512;
constexpr static int FinalizeWeightsBlockSize = 256;

//These numbers are not at all optimized,
//just going from rough similarity to TAC operations
//(which themselves are not that optimised
// since they were last tested on a previous version...)

#if defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ > 350
  #if CUDART_VERSION >= 12000
    #define CAN_USE_TAIL_LAUNCH 1
  #else
    #define CAN_USE_TAIL_LAUNCH 0
  #endif
#elif defined(__CUDA_ARCH__)
  #error "CUDA compute capability at least 3.5 is needed so we can have dynamic parallelism!"
#endif

namespace TASHacks
//We will (ab)use the cluster info to hold the original cluster reference,
//shared cluster cells list and continue flag,
//as the cluster indices fit in an int (in fact, they're 16 bits)
//and the cluster properties are only overridden later.
//Some pointer trickery, but, given that CUDA allocations
//are, in their essence, casted from void * as in C,
//this should be safe, even more since int and float are both 32-bit.
{
  static __device__ int get_original_cluster_table_entry(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                         const int index)
  {
    return __float_as_int(clusters_arr->clusterEnergy[index]);
  }

  static __device__ void set_original_cluster_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                          const int index, const int new_value)
  {
    clusters_arr->clusterEnergy[index] = __int_as_float(new_value);
  }

  /*
  static __host__ __device__ int * get_original_cluster_table_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                                      const int index)
  {
    void * v_ptr = &(clusters_arr->clusterEnergy[index]);
    return (int *) v_ptr;
  }
  */


  static __device__ int get_continue_flag(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return __float_as_int(clusters_arr->clusterPhi[NMaxClusters - 1]);
  }

  static __device__ void set_continue_flag(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int value)
  {
    clusters_arr->clusterPhi[NMaxClusters - 1] = __int_as_float(value);
  }

  static __host__ __device__ int * get_continue_flag_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    void * v_ptr = &(clusters_arr->clusterPhi[NMaxClusters - 1]);
    return (int *) v_ptr;
  }


  //Extra pairs of neighbours used in checking for maxima
  //but not to actually grow the clusters.
  static __device__ int get_num_extra_neighs(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return __float_as_int(clusters_arr->clusterPhi[NMaxClusters - 2]);
  }

  /*
  static __device__ void set_num_extra_neighs(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int value)
  {
    clusters_arr->clusterPhi[NMaxClusters - 2] = __int_as_float(value);
  }
  */

  static __host__ __device__ int * get_num_extra_neighs_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    void * v_ptr = &(clusters_arr->clusterPhi[NMaxClusters - 2]);
    return (int *) v_ptr;
  }


  static __device__ int get_num_extra_reverse_neighs(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return __float_as_int(clusters_arr->clusterPhi[NMaxClusters - 3]);
  }

  /*
  static __device__ void set_num_extra_reverse_neighs(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int value)
  {
    clusters_arr->clusterPhi[NMaxClusters - 3] = __int_as_float(value);
  }
  */

  static __host__ __device__ int * get_num_extra_reverse_neighs_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    void * v_ptr = &(clusters_arr->clusterPhi[NMaxClusters - 3]);
    return (int *) v_ptr;
  }



  //We must do this this way because there are more cells than NMaxClusters,
  //so we spill over to the next cluster properties (Eta and the first ~third of Phi)
  static __host__ __device__ int * get_cell_to_cluster_table_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                                     const int index)
  {
    void * v_ptr = &(clusters_arr->clusterEt[0]);
    return ((int *) v_ptr) + index;
  }

  static __device__ int get_cell_to_cluster_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                        const int index)
  {
    return *(get_cell_to_cluster_table_address(clusters_arr, index));
  }

  static __device__ void set_cell_to_cluster_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                         const int index, const int new_value)
  {
    *(get_cell_to_cluster_table_address(clusters_arr, index)) = new_value;
  }

  static __device__ int get_reset_counter(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return __float_as_int(clusters_arr->clusterPhi[NMaxClusters - 4]);
  }

  static __device__ void set_reset_counter(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int value)
  {
    clusters_arr->clusterPhi[NMaxClusters - 4] = __int_as_float(value);
  }

  static __host__ __device__ int * get_reset_counter_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    void * v_ptr = &(clusters_arr->clusterPhi[NMaxClusters - 4]);
    return (int *) v_ptr;
  }

  //This next table is used in the secondary maxima invalidation
  //to be able to iterate in both directions simultaneously using
  //the main and secondary arrays, while storing the intermediate
  //cell assignments here in a reversible form...

  static __host__ __device__ int * get_secondary_restore_table_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                                       const int index)
  {
    void * v_ptr = &(clusters_arr->clusterEt[0]);
    return ((int *) v_ptr) + index;
  }

  static __device__ int get_secondary_restore_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                          const int index)
  {
    return *(get_secondary_restore_table_address(clusters_arr, index));
  }

  static __device__ void set_secondary_restore_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                           const int index, const int new_value)
  {
    *(get_secondary_restore_table_address(clusters_arr, index)) = new_value;
  }

#if !CAN_USE_TAIL_LAUNCH

  static __device__ int get_stop_flag(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return __float_as_int(clusters_arr->clusterPhi[NMaxClusters - 5]);
  }

  static __device__ void set_stop_flag(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int value)
  {
    clusters_arr->clusterPhi[NMaxClusters - 5] = __int_as_float(value);
  }

  static __host__ __device__ int * get_stop_flag_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    void * v_ptr = &(clusters_arr->clusterPhi[NMaxClusters - 5]);
    return (int *) v_ptr;
  }
#endif

}

/******************************************************************************************
 * Determine the same-cluster neighbours of the cells and fill the pairs list accordingly.
 ******************************************************************************************/


static __global__
void fillNeighboursFirstKernel( Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < NCaloCells)
    {
      const ClusterTag this_tag = cell_state_arr->clusterTag[index];

      if (this_tag.is_part_of_cluster())
        {
          const bool is_limited = geometry->neighbours.has_limited_neighbours(index, opts->limit_HECIW_and_FCal_neighs, opts->limit_PS_neighs);
          //The cells that have limited neighbours, for the split cluster growing part.
          //WARNING: the CPU version of the code does not limit PS neighbours ever, but we give additional freedom
          //         (even if it is disabled by default).

          int neighbours[NMaxNeighbours], good_neighbours[NMaxNeighbours];

          const unsigned int limited_flags = LArNeighbours::neighbourOption::nextInSamp;

          const unsigned int neighbour_flags = LArNeighbours::neighbourOption::nextSuperCalo;
          //We will add the rest of the maxima exclusion neighbours
          //in the second fill neighbours kernel, since they must be
          //iterated over separately for everything to work properly.


          const int num_limited = geometry->neighbours.get_neighbours_with_option(limited_flags & opts->neighbour_options, index, neighbours);

          const int num_no_secondary_max = geometry->neighbours.get_neighbours_with_option( neighbour_flags & opts->neighbour_options,
                                                                                            index, &(neighbours[num_limited])           ) + num_limited;
          //The limited also belong to the secondary maximum exclusion.

          const int num_others = geometry->neighbours.get_neighbours_with_option( ( ~(neighbour_flags | limited_flags) ) & opts->neighbour_options,
                                                                                  index, &(neighbours[num_no_secondary_max])                     );

          const int num_total_neighs = num_no_secondary_max + num_others;

          int num_good_neighs = 0;

          for (int i = 0; (i < num_limited || !is_limited) && i < num_total_neighs; ++i)
            {
              const int neigh_ID = neighbours[i];
              const ClusterTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];
              if (neigh_tag.is_part_of_cluster() && this_tag.cluster_index() == neigh_tag.cluster_index())
                {
                  good_neighbours[num_good_neighs] = neigh_ID;
                  ++num_good_neighs;
                }
            }
          if (num_good_neighs > 0)
            {
              const int n = atomicAdd(&(neighbour_pairs->number), num_good_neighs);
              for (int i = 0; i < num_good_neighs; ++i)
                {
                  neighbour_pairs->cellID[n + i] = good_neighbours[i];
                  neighbour_pairs->neighbourID[n + i] = index;
                }
            }

          if (num_no_secondary_max > 0)
            //Don't forget that secondary maxima invalidation
            //occurs regardless of cluster...
            {
              const int n = atomicAdd(&(neighbour_pairs->reverse_number), num_no_secondary_max);
              const int real_start = NMaxPairs - n - num_no_secondary_max;
              for (int i = 0; i < num_no_secondary_max; ++i)
                {
                  neighbour_pairs->cellID[real_start + i] = neighbours[i];
                  neighbour_pairs->neighbourID[real_start + i] = index;
                }
            }
          temporaries->get_extra_array<int>(index) = num_good_neighs;
        }
      else
        {
          int neighbours[NMaxNeighbours];

          const unsigned int neighbour_flags = LArNeighbours::neighbourOption::nextInSamp |
                                               LArNeighbours::neighbourOption::nextSuperCalo;

          const int num_neighs = geometry->neighbours.get_neighbours_with_option(neighbour_flags & opts->neighbour_options, index, neighbours);

          if (num_neighs > 0)
            //Don't forget that secondary maxima invalidation
            //occurs regardless of cluster...
            {
              const int n = atomicAdd(&(neighbour_pairs->reverse_number), num_neighs);
              const int real_start = NMaxPairs - n - num_neighs;
              for (int i = 0; i < num_neighs; ++i)
                {
                  neighbour_pairs->cellID[real_start + i] = neighbours[i];
                  neighbour_pairs->neighbourID[real_start + i] = index;
                }
            }

          temporaries->get_extra_array<int>(index) = -NMaxNeighbours;
        }
    }
}


static __global__
void fillNeighboursSecondKernel( Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                 Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                 const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                 const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                 const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                 const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts )
//Local maxima are checked even with the neighbourhood relations
//that end up otherwise being excluded in the cells that are limited
//to `nextInSample`. We add the relevant pairs here after the end of the normal pairs
//and store their numbers on the appropriate temporary.
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < NCaloCells)
    {
      const ClusterTag this_tag = cell_state_arr->clusterTag[index];

      int neighbours[NMaxNeighbours];

      if (this_tag.is_part_of_cluster())
        {
          if ( geometry->neighbours.has_limited_neighbours(index, opts->limit_HECIW_and_FCal_neighs, opts->limit_PS_neighs) )
            //WARNING: the CPU version of the code does not limit PS neighbours ever, but we give additional freedom
            //         (even if it is disabled by default).
            {
              int good_neighbours[NMaxNeighbours];
              const unsigned int limited_flags = LArNeighbours::neighbourOption::nextInSamp;

              const int num_total_neighs = geometry->neighbours.get_neighbours_with_option((~limited_flags) & opts->neighbour_options, index, neighbours);
              int num_good_neighs = 0;

              for (int i = 0; i < num_total_neighs; ++i)
                {
                  const int neigh_ID = neighbours[i];
                  const ClusterTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];
                  if (neigh_tag.is_part_of_cluster() && this_tag.cluster_index() == neigh_tag.cluster_index())
                    {
                      good_neighbours[num_good_neighs] = neigh_ID;
                      ++num_good_neighs;
                    }
                }

              if (num_good_neighs > 0)
                {
                  const int n = atomicAdd(TASHacks::get_num_extra_neighs_address(clusters_arr), num_good_neighs);
                  const int start = neighbour_pairs->number + n;
                  for (int i = 0; i < num_good_neighs; ++i)
                    {
                      neighbour_pairs->cellID[start + i] = good_neighbours[i];
                      neighbour_pairs->neighbourID[start + i] = index;
                    }
                }
              temporaries->get_extra_array<int>(index) += num_good_neighs;
            }

        }

      const unsigned int no_max_flags = LArNeighbours::neighbourOption::prevInSamp |
                                        LArNeighbours::neighbourOption::prevSuperCalo;
      const int num_neighs = geometry->neighbours.get_neighbours_with_option(no_max_flags & opts->neighbour_options, index, neighbours);

      if (num_neighs > 0)
        //Don't forget that secondary maxima invalidation
        //occurs regardless of cluster...
        {
          const int n = atomicAdd(TASHacks::get_num_extra_reverse_neighs_address(clusters_arr), num_neighs);
          const int real_start = NMaxPairs - n - num_neighs - neighbour_pairs->reverse_number;
          for (int i = 0; i < num_neighs; ++i)
            {
              neighbour_pairs->cellID[real_start + i] = neighbours[i];
              neighbour_pairs->neighbourID[real_start + i] = index;
            }
        }
    }
}

void fillNeighbours(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temps,
                    const ConstantDataHolder & instance_data, const TASOptionsHolder & options, const bool synchronize,
                    CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  hipMemsetAsync(&(holder.m_pairs_dev->number), 0, sizeof(holder.m_pairs_dev->number), stream_to_use);
  hipMemsetAsync(&(holder.m_pairs_dev->reverse_number), 0, sizeof(holder.m_pairs_dev->reverse_number), stream_to_use);
  hipMemsetAsync(TASHacks::get_num_extra_neighs_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
  hipMemsetAsync(TASHacks::get_num_extra_reverse_neighs_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);

  const int i_dimBlock1 = FillNeighboursFirstBlockSize;
  const int i_dimGrid1 = Helpers::int_ceil_div(NCaloCells, i_dimBlock1);
  const dim3 dimBlock1(i_dimBlock1, 1, 1);
  const dim3 dimGrid1(i_dimGrid1, 1, 1);

  const int i_dimBlock2 = FillNeighboursSecondBlockSize;
  const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells, i_dimBlock2);
  const dim3 dimBlock2(i_dimBlock2, 1, 1);
  const dim3 dimGrid2(i_dimGrid2, 1, 1);

  fillNeighboursFirstKernel <<< dimGrid1, dimBlock1, 0, stream_to_use>>>(holder.m_pairs_dev, temps, holder.m_cell_state_dev,
                                                                         instance_data.m_geometry_dev, options.m_options_dev);

  fillNeighboursSecondKernel <<< dimGrid2, dimBlock2, 0, stream_to_use>>>(holder.m_pairs_dev, temps, holder.m_clusters_dev,
                                                                          holder.m_cell_state_dev, instance_data.m_geometry_dev, options.m_options_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/******************************************************************************************
 * Determine the local maxima and initialize the cell arrays appropriately.
 ******************************************************************************************/

static __global__
void countInferiorNeighsKernel( Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                const int num_normal_pairs,
                                const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < num_normal_pairs)
    {
      const int this_ID = neighbour_pairs->cellID[index];
      const int neigh_ID = neighbour_pairs->neighbourID[index];

      const int this_sampling = geometry->caloSample[this_ID];
      const int neigh_sampling = geometry->caloSample[neigh_ID];

      float this_energy = 0.f, neigh_energy = 0.f;

      if (!cell_info_arr->is_bad(*geometry, this_ID, opts->treat_L1_predicted_as_good) && opts->uses_sampling(this_sampling))
        {
          this_energy = cell_info_arr->energy[this_ID];
          if (opts->use_absolute_energy)
            {
              this_energy = fabsf(this_energy);
            }
          else if (this_energy <= 0.f)
            {
              this_energy = 0.f;
            }
        }

      if (!cell_info_arr->is_bad(*geometry, neigh_ID, opts->treat_L1_predicted_as_good) && opts->uses_sampling(neigh_sampling))
        {
          neigh_energy = cell_info_arr->energy[neigh_ID];
          if (opts->use_absolute_energy)
            {
              neigh_energy = fabsf(neigh_energy);
            }
          else if (neigh_energy <= 0.f)
            {
              neigh_energy = 0.f;
            }
        }

      bool is_max_neig = neigh_energy > this_energy;

      if (opts->uses_primary_sampling(neigh_sampling))
        {
          if (!opts->uses_primary_sampling(this_sampling) && opts->uses_secondary_sampling(this_sampling))
            {
              is_max_neig = true;
            }
        }

      if (!is_max_neig)
        {
          temporaries->get_extra_array<int>(neigh_ID) = -NCaloCells;
          //No need to count, just to invalidate!
        }
    }
}

static __global__
void countNeighsDeferrerKernel(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                               const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                               const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                               const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                               const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                               const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int full_pairs_number = neighbour_pairs->number + TASHacks::get_num_extra_neighs(clusters_arr);

      const int i_dimBlock1 = CountInferiorNeighsBlockSize;
      const int i_dimGrid1 = Helpers::int_ceil_div(full_pairs_number, i_dimBlock1);
      const dim3 dimBlock1(i_dimBlock1, 1, 1);
      const dim3 dimGrid1(i_dimGrid1, 1, 1);
#if CAN_USE_TAIL_LAUNCH
      countInferiorNeighsKernel <<< dimGrid1, dimBlock1, 0, cudaStreamTailLaunch>>>(temporaries, cell_info_arr, neighbour_pairs, full_pairs_number, geometry, opts);
#else
      countInferiorNeighsKernel <<< dimGrid1, dimBlock1>>>(temporaries, cell_info_arr, neighbour_pairs, full_pairs_number, geometry, opts);
#endif
    }
}

static __global__
void findLocalMaximaKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                            Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                            const Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                            const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                            const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                            const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < NCaloCells)
    {
      const ClusterTag this_tag = cell_state_arr->clusterTag[index];

      if (this_tag.is_part_of_cluster())
        {
          const int this_sampling = geometry->caloSample[index];

          float cell_energy = 0.f;
          const float raw_cell_energy = cell_info_arr->energy[index];

          if (!cell_info_arr->is_bad(*geometry, index, opts->treat_L1_predicted_as_good) && opts->uses_sampling(this_sampling))
            {
              cell_energy = raw_cell_energy;
              if (opts->use_absolute_energy)
                {
                  cell_energy = fabsf(cell_energy);
                }
              else if (cell_energy <= 0.f)
                {
                  cell_energy = 0.f;
                }
            }

          const int num_neighs = temporaries->get_extra_array<int>(index);

          bool is_primary = false, is_maximum = false;

          if (num_neighs >= 0 && num_neighs >= opts->min_num_cells && cell_energy >= opts->min_maximum_energy)
            {
              if (opts->uses_primary_sampling(this_sampling))
                {
                  is_maximum = true;
                  is_primary = true;
                }
              else if (opts->uses_secondary_sampling(this_sampling))
                {
                  is_maximum = true;
                  is_primary = false;
                }
            }

          if (is_maximum)
            {
              const int original_cluster = this_tag.cluster_index();
              const int new_cluster = atomicAdd(&(clusters_arr->number), 1);


              const TASTag new_tag = TASTag::make_maximum_tag(new_cluster, __float_as_uint(raw_cell_energy), is_primary);

              cell_state_arr->clusterTag[index] = new_tag;

              clusters_arr->seedCellID[new_cluster] = index;

              clusters_arr->seedCellID[original_cluster] = -1;

              TASHacks::set_original_cluster_table_entry(clusters_arr, new_cluster, original_cluster);

            }
          else
            {
              const int this_cluster = this_tag.cluster_index();

              cell_state_arr->clusterTag[index] = TASTag::make_cluster_cell_tag(this_cluster);
            }
        }
      else
        {
          cell_state_arr->clusterTag[index] = TASTag::make_invalid_tag();
        }
    }
}

static __global__
void fixClustersWithoutMaximaKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                     Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                     Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < NCaloCells)
    {
      TASTag this_tag = cell_state_arr->clusterTag[index];

      if (this_tag.is_part_of_splitter_cluster())
        {
          temporaries->secondaryArray[index] = this_tag;
        }
      else if (this_tag.is_valid())
        //It'll be part of an original cluster, given how we've assigned the tags.
        {
          const int cluster_index = this_tag.index();

          if (clusters_arr->seedCellID[cluster_index] >= 0)
            //This means the cluster is not split.
            {
              TASTag new_tag = TASTag::make_original_cluster_tag(cluster_index);
              cell_state_arr->clusterTag[index] = new_tag;
              temporaries->secondaryArray[index] = new_tag;
              TASHacks::set_original_cluster_table_entry(clusters_arr, cluster_index, cluster_index);
            }
          else
            //This means the tag belongs to a cluster to be split
            {
              temporaries->secondaryArray[index] = this_tag;
            }
        }
      else
        {
          temporaries->secondaryArray[index] = TASTag::make_invalid_tag();
        }
    }

}

void findLocalMaxima(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temps,
                     const ConstantDataHolder & instance_data, const TASOptionsHolder & options, const bool synchronize,
                     CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  const int i_dimBlock1 = FindLocalMaximaBlockSize;
  const int i_dimGrid1 = Helpers::int_ceil_div(NCaloCells, i_dimBlock1);
  const dim3 dimBlock1(i_dimBlock1, 1, 1);
  const dim3 dimGrid1(i_dimGrid1, 1, 1);

  const int i_dimBlock2 = FixClustersWithoutMaximaBlockSize;
  const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells, i_dimBlock2);
  const dim3 dimBlock2(i_dimBlock2, 1, 1);
  const dim3 dimGrid2(i_dimGrid2, 1, 1);

  countNeighsDeferrerKernel <<< 1, 1, 0, stream_to_use>>>(temps, holder.m_cell_info_dev, holder.m_clusters_dev,
                                                          holder.m_pairs_dev, instance_data.m_geometry_dev, options.m_options_dev);


  findLocalMaximaKernel <<< dimGrid1, dimBlock1, 0, stream_to_use>>>(holder.m_cell_state_dev, holder.m_clusters_dev,
                                                                     temps, holder.m_cell_info_dev,
                                                                     instance_data.m_geometry_dev, options.m_options_dev);

  fixClustersWithoutMaximaKernel <<< dimGrid2, dimBlock2, 0, stream_to_use>>>(holder.m_cell_state_dev, temps, holder.m_clusters_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}

/*****************************************************************************
 * Delete secondary maxima according to the criteria on the CPU version.
 ******************************************************************************/

namespace
{
  constexpr int primary_cluster_mark  = 0x40000000;
  constexpr int original_cluster_mark = 0x20000000;
  constexpr int part_of_cluster_mark  = 0x10000000;
  constexpr int invalid_cell_value    = ~( primary_cluster_mark  |
                                           original_cluster_mark |
                                           part_of_cluster_mark    );
}

__global__ static
void prepareArraysForSecondaryMaxima(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                     Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                     Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr                    )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      TASTag this_tag = cell_state_arr->clusterTag[index];

      if (this_tag.is_part_of_original_cluster())
        {
          TASHacks::set_secondary_restore_table_entry(clusters_arr, index, this_tag.index() | original_cluster_mark);
          this_tag = TASTag::make_invalid_tag();
          //We can propagate freely through original clusters!
        }
      else if (this_tag.is_part_of_splitter_cluster())
        {
          if (this_tag.is_secondary())
            {
              TASHacks::set_secondary_restore_table_entry(clusters_arr, index, this_tag.index());
              this_tag = this_tag.update_index(index);
              //To get the correct ordering.
            }
          else
            {
              TASHacks::set_secondary_restore_table_entry(clusters_arr, index, this_tag.index() | primary_cluster_mark);
              this_tag = TASTag::secondary_maxima_eliminator();
            }
        }
      else if (this_tag.is_non_assigned_part_of_split_cluster())
        {
          TASHacks::set_secondary_restore_table_entry(clusters_arr, index, this_tag.index() | part_of_cluster_mark);
          this_tag = TASTag::make_invalid_tag();
        }
      else
        {
          TASHacks::set_secondary_restore_table_entry(clusters_arr, index, invalid_cell_value);
          this_tag = TASTag::make_invalid_tag();
        }

      cell_state_arr->clusterTag[index] = this_tag;
      temporaries->secondaryArray[index] = this_tag;
    }
}

__global__ static
void checkForMaximaExclusionTermination( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                         Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                         const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs);

__global__ static
void propagateForMaximaExclusion( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                  Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                  Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                  const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                  const int pair_start,
                                  const int pair_switch,
                                  const int pair_number)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < pair_number)
    {
      const int this_ID = neighbour_pairs->cellID[pair_start + index];
      const int neigh_ID = neighbour_pairs->neighbourID[pair_start + index];

      tag_type * array = ( index < pair_switch ?
                           cell_state_arr->clusterTag :
                           temporaries->secondaryArray  );

      const TASTag this_tag = array[this_ID];

      if (this_tag.is_secondary_maxima_eliminator() || this_tag.is_secondary_maximum_seed())
        {
          if (atomicMax(&(array[neigh_ID]), this_tag) < this_tag)
            {
              TASHacks::set_continue_flag(clusters_arr, 1);
            }
        }
    }
#if CAN_USE_TAIL_LAUNCH
  else if (index == pair_number)
    {
      checkForMaximaExclusionTermination <<< 1, 1, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs);
    }
#endif
}

__global__ static
void checkForMaximaExclusionTermination( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                         Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                         const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      if (TASHacks::get_continue_flag(clusters_arr))
        {
          TASHacks::set_continue_flag(clusters_arr, 0);
#if CAN_USE_TAIL_LAUNCH
          const int reverse_pairs_number = neighbour_pairs->reverse_number;
          const int reverse_pairs_start = NMaxPairs - reverse_pairs_number;

          const int extra_reverse_pairs_number = TASHacks::get_num_extra_reverse_neighs(clusters_arr);

          const int total_pairs_start = reverse_pairs_start - extra_reverse_pairs_number;

          const int total_pairs_number = reverse_pairs_number + extra_reverse_pairs_number;

          const int i_dimBlock1 = ExcludeMaximaPropagationBlockSize;
          const int i_dimGrid1 = Helpers::int_ceil_div(total_pairs_number + 1, i_dimBlock1);
          const dim3 dimBlock1(i_dimBlock1, 1, 1);
          const dim3 dimGrid1(i_dimGrid1, 1, 1);
          propagateForMaximaExclusion <<< dimGrid1, dimBlock1>>>(cell_state_arr, temporaries,
                                                                 clusters_arr, neighbour_pairs,
                                                                 total_pairs_start, extra_reverse_pairs_number,
                                                                 total_pairs_number);

#endif
        }
#if !CAN_USE_TAIL_LAUNCH
      else /*if (!TASHacks::get_continue_flag(clusters_arr))*/
        {
          TASHacks::set_stop_flag(clusters_arr, 1);
        }
#endif
    }
}

__global__ static
void excludeSecondaryMaximaKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                   Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                   Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                   const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                   const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int reverse_pairs_number = neighbour_pairs->reverse_number;
      const int reverse_pairs_start = NMaxPairs - reverse_pairs_number;

      const int extra_reverse_pairs_number = TASHacks::get_num_extra_reverse_neighs(clusters_arr);

      const int total_pairs_start = reverse_pairs_start - extra_reverse_pairs_number;

      const int total_pairs_number = reverse_pairs_number + extra_reverse_pairs_number;

      const int i_dimBlock1 = ExcludeMaximaPropagationBlockSize;
      const int i_dimGrid1 = Helpers::int_ceil_div(total_pairs_number + 1, i_dimBlock1);
      const dim3 dimBlock1(i_dimBlock1, 1, 1);
      const dim3 dimGrid1(i_dimGrid1, 1, 1);

#if CAN_USE_TAIL_LAUNCH
      propagateForMaximaExclusion <<< dimGrid1, dimBlock1>>>(cell_state_arr, temporaries,
                                                             clusters_arr, neighbour_pairs,
                                                             total_pairs_start, extra_reverse_pairs_number,
                                                             total_pairs_number);
#else
      while (!TASHacks::get_stop_flag(clusters_arr))
        {
          propagateForMaximaExclusion <<< dimGrid1, dimBlock1>>>(cell_state_arr, temporaries,
                                                                 clusters_arr, neighbour_pairs,
                                                                 total_pairs_start, extra_reverse_pairs_number,
                                                                 total_pairs_number);
          checkForMaximaExclusionTermination <<< 1, 1>>> (cell_state_arr, temporaries,
                                                          clusters_arr, neighbour_pairs);
          //++counter;
        }

      //printf("COUNTS: %16d\n", counter);
#endif
    }
}



__global__ static
void resetAndCleanUpSecondaries(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const TASTag tag_one = cell_state_arr->clusterTag[index];
      const TASTag tag_two = temporaries->secondaryArray[index];
      const int prev_state = TASHacks::get_secondary_restore_table_entry(clusters_arr, index);
      const int prev_index = prev_state & 0xFFFF;
      const float  energy  = cell_info_arr->energy[index];

      TASTag new_tag = 0;

      if (prev_state == invalid_cell_value)
        {
          new_tag = TASTag::make_invalid_tag();
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, -1);
        }
      else if (prev_state == prev_index)
        //is a secondary cluster.
        {
          if (tag_one.index() != index || tag_two.index() != index)
            //The tag got replaced with that of a more energetic secondary maximum
            //or a primary maximum (being a secondary maxima eliminator,
            //which also has an index larger than any cell index...)
            {
              clusters_arr->seedCellID[prev_index] = -1;
              const int old_index = TASHacks::get_original_cluster_table_entry(clusters_arr, prev_index);
              new_tag = TASTag::make_cluster_cell_tag(old_index);
              TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, old_index | (old_index << 16));
              //This actually is the same as the restore table,
              //just different functions for semantic clarity...
            }
          else
            {
              //clusters_arr->seedCellID[prev_index] = index;
              new_tag = TASTag::make_maximum_tag(index, __float_as_uint(energy), true);
              //No further distinction between primaries and secondaries.
              TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, prev_index);
            }
        }
      else if (prev_state & original_cluster_mark)
        {
          new_tag = TASTag::make_original_cluster_tag(prev_index);
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, prev_index | (prev_index << 16));
        }
      else if (prev_state & part_of_cluster_mark)
        {
          new_tag = TASTag::make_cluster_cell_tag(prev_index);
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, prev_index | (prev_index << 16));
        }
      else if (prev_state & primary_cluster_mark)
        {
          new_tag = TASTag::make_maximum_tag(index, __float_as_uint(energy), true);
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, prev_index);
        }
      else
        {
          new_tag = TASTag::make_invalid_tag();
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, -1);
        }

      cell_state_arr->clusterTag[index] = new_tag;
      temporaries->secondaryArray[index] = new_tag;
    }
}

void excludeSecondaryMaxima(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temps,
                            const ConstantDataHolder & instance_data, const TASOptionsHolder & options, const bool synchronize,
                            CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  const int i_dimBlock1 = PrepareArrayForSecondaryMaximaBlockSize;
  const int i_dimGrid1 = Helpers::int_ceil_div(NCaloCells, i_dimBlock1);
  const dim3 dimBlock1(i_dimBlock1, 1, 1);
  const dim3 dimGrid1(i_dimGrid1, 1, 1);

  const int i_dimBlock2 = ResetAndCleanSecondariesBlockSize;
  const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells, i_dimBlock2);
  const dim3 dimBlock2(i_dimBlock2, 1, 1);
  const dim3 dimGrid2(i_dimGrid2, 1, 1);

  prepareArraysForSecondaryMaxima <<< dimGrid1, dimBlock1>>>(holder.m_cell_state_dev, temps, holder.m_clusters_dev);
  if (options.m_options->valid_sampling_secondary != 0)
    {
      hipMemsetAsync(TASHacks::get_continue_flag_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
#if !CAN_USE_TAIL_LAUNCH
      hipMemsetAsync(TASHacks::get_stop_flag_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
#endif
      excludeSecondaryMaximaKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_cell_state_dev, holder.m_clusters_dev, temps, holder.m_pairs_dev, holder.m_cell_info_dev);
    }
  resetAndCleanUpSecondaries <<< dimGrid2, dimBlock2, 0, stream_to_use>>>(holder.m_cell_state_dev, temps, holder.m_clusters_dev, holder.m_cell_info_dev);
  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/******************************************************************************************
 * Propagate the new tags and create the final clusters.
 ******************************************************************************************/
__global__ static
void handleSplitterIndexChangesKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                       Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                       Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                       const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                       const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                       const bool share_cells                                                       );

__global__ static
void handleSplitterTagChangesKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                     Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                     Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                     const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                     const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                     const bool share_cells                                                        );

__global__ static
void checkForTagPropagationTermination( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                        Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                        Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                        const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                        const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                        const bool share_cells                                                       );

__global__ static
void propagateSplitterTagsKernel( const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                  Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                  Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                  const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                  const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                  const int pair_number, const bool share_cells                                 )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < pair_number)
    {
      const int this_ID = neighbour_pairs->cellID[index];
      const int neigh_ID = neighbour_pairs->neighbourID[index];

      const TASTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];

      if (!neigh_tag.is_part_of_splitter_cluster())
        {
          return;
        }

      TASTag prop_tag = neigh_tag.propagate();

      if (neigh_tag.is_shared() && !neigh_tag.is_primary() && neigh_tag.counter() > 0x7FF)
        {
          prop_tag = prop_tag.update_counter(0x7FF);
          //Shared cells after the original ones
          //are not ordered by the propagation step
          //of the original shared cell.
          //Assuming less than 2^11 = 2048 propagation steps
          //before making a shared cell seems safe-ish?
        }

      const TASTag old_tag = cell_state_arr->clusterTag[this_ID];
      if (share_cells && !neigh_tag.is_shared() && old_tag.is_part_of_splitter_cluster() && !old_tag.is_shared() && !old_tag.is_primary())
        {
          const int old_count = old_tag.counter();
          const int new_count = prop_tag.counter();
          const int old_cell = old_tag.index();
          const int new_cell = prop_tag.index();
          if (old_count == new_count && old_cell != new_cell)
            //Note that, in the CPU implementation,
            //cells are only shared if they are in the to-grow list.
            {
              const int old_index = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, old_cell);
              const int new_index = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, new_cell);
              if (old_index != new_index)
                {
                  prop_tag = old_tag.prepare_for_sharing(prop_tag);
                  atomicMax(TASHacks::get_reset_counter_address(clusters_arr), old_count);
                }
            }
        }

      atomicMax(&(temporaries->secondaryArray[this_ID]), prop_tag);
    }
#if CAN_USE_TAIL_LAUNCH
  else if (index == pair_number)
    {
      const int i_dimBlock = HandleSplitIndexChangesBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(NCaloCells + 1, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);

      handleSplitterIndexChangesKernel <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, cell_info_arr, neighbour_pairs, share_cells);
    }
#endif
}

__global__ static
void handleSplitterIndexChangesKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                       Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                       Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                       const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                       const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                       const bool share_cells                                                             )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      TASTag old_tag = cell_state_arr->clusterTag[index];
      TASTag new_tag = temporaries->secondaryArray[index];

      if (!new_tag.is_part_of_splitter_cluster() || new_tag.counter() == TASTag::max_counter())
        {
          return;
        }

      const uint32_t new_cluster_index = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, new_tag.index());
      const int desired_counter = TASHacks::get_reset_counter(clusters_arr);

      if ( new_tag.counter() < desired_counter || (old_tag.is_part_of_splitter_cluster() && old_tag.counter() < desired_counter) )
        {
          const int original_cluster_index = TASHacks::get_original_cluster_table_entry(clusters_arr, new_cluster_index & 0xFFFFU);
          new_tag = TASTag::make_cluster_cell_tag(original_cluster_index);
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, original_cluster_index | (original_cluster_index << 16));
          TASHacks::set_continue_flag(clusters_arr, 1);
          cell_state_arr->clusterTag[index] = new_tag;
          temporaries->secondaryArray[index] = new_tag;
        }
      else if (!new_tag.is_primary())
        {
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, new_cluster_index);
        }
    }
#if CAN_USE_TAIL_LAUNCH
  else if (index == NCaloCells)
    {
      const int i_dimBlock = HandleSplitTagChangesBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(NCaloCells + 1, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);
      handleSplitterTagChangesKernel <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, cell_info_arr, neighbour_pairs, share_cells);
    }
#endif
}

__global__ static
void handleSplitterTagChangesKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                     Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                     Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                     const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                     const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                     const bool share_cells                                                             )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      TASTag old_tag = cell_state_arr->clusterTag[index];
      TASTag new_tag = temporaries->secondaryArray[index];

      if (!new_tag.is_part_of_splitter_cluster() || old_tag == new_tag)
        {
          return;
        }

      const uint32_t old_cluster_index = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, old_tag.index());
      const uint32_t new_cluster_index = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, new_tag.index());

      if (old_cluster_index == new_cluster_index)
        {
          const float cell_energy = cell_info_arr->energy[index];
          new_tag = new_tag.update_cell(index, __float_as_uint(cell_energy));
          if (new_tag != old_tag)
            {
              TASHacks::set_continue_flag(clusters_arr, 1);
            }
          cell_state_arr->clusterTag[index] = new_tag;
          temporaries->secondaryArray[index] = new_tag;
          return;
        }

      if ( old_tag.is_part_of_splitter_cluster() && !old_tag.is_shared() &&
           new_tag.is_shared() && new_tag.is_primary()                        )
        {
          new_tag = new_tag.update_counter(old_tag.counter() + 1);
          const int min_index = min(new_cluster_index, old_cluster_index) & 0xFFFF;
          const int max_index = max(new_cluster_index, old_cluster_index) & 0xFFFF;
          TASHacks::set_cell_to_cluster_table_entry(clusters_arr, index, (max_index << 16) | min_index);
          const float cell_energy = cell_info_arr->energy[index];
          new_tag = new_tag.update_cell(index, __float_as_uint(cell_energy));
        }
      else
        {
          const float cell_energy = cell_info_arr->energy[index];
          new_tag = new_tag.update_cell(index, __float_as_uint(cell_energy));
        }

      TASHacks::set_continue_flag(clusters_arr, 1);
      cell_state_arr->clusterTag[index] = new_tag;
      temporaries->secondaryArray[index] = new_tag;
    }
#if CAN_USE_TAIL_LAUNCH
  else if (index == NCaloCells)
    {
      checkForTagPropagationTermination <<< 1, 1, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, cell_info_arr, neighbour_pairs, share_cells);
    }
#endif
}


__global__ static
void checkForTagPropagationTermination( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                        Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                        Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                        const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                        const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                        const bool share_cells                                                             )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      TASHacks::set_reset_counter(clusters_arr, 0);
      if (TASHacks::get_continue_flag(clusters_arr))
        {
          TASHacks::set_continue_flag(clusters_arr, 0);
#if CAN_USE_TAIL_LAUNCH
          const int pairs_number = neighbour_pairs->number;

          const int i_dimBlock = PropagateSplitTagsBlockSize;
          const int i_dimGrid = Helpers::int_ceil_div(pairs_number + 1, i_dimBlock);
          const dim3 dimBlock(i_dimBlock, 1, 1);
          const dim3 dimGrid(i_dimGrid, 1, 1);

          propagateSplitterTagsKernel <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr,
                                                                                        cell_info_arr, neighbour_pairs, pairs_number, share_cells);
#endif
        }
#if !CAN_USE_TAIL_LAUNCH
      else /*if (!TASHacks::get_continue_flag(clusters_arr))*/
        {
          TASHacks::set_stop_flag(clusters_arr, 1);
        }
#endif
    }
}


__global__ static
void splitClusterTagPropagationKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                       Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                       Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                       const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                       const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                                       const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int pairs_number = neighbour_pairs->number;

      const int i_dimBlock1 = PropagateSplitTagsBlockSize;
      const int i_dimGrid1 = Helpers::int_ceil_div(pairs_number + 1, i_dimBlock1);
      const dim3 dimBlock1(i_dimBlock1, 1, 1);
      const dim3 dimGrid1(i_dimGrid1, 1, 1);
#if CAN_USE_TAIL_LAUNCH

#else
      const int i_dimBlock2 = HandleSplitIndexChangesBlockSize;
      const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells, i_dimBlock2);
      const dim3 dimBlock2(i_dimBlock2, 1, 1);
      const dim3 dimGrid2(i_dimGrid2, 1, 1);

      const int i_dimBlock3 = HandleSplitTagChangesBlockSize;
      const int i_dimGrid3 = Helpers::int_ceil_div(NCaloCells, i_dimBlock3);
      const dim3 dimBlock3(i_dimBlock3, 1, 1);
      const dim3 dimGrid3(i_dimGrid3, 1, 1);

      //int counter = 0;

      const bool share_cells = opts->share_border_cells;

      while (!TASHacks::get_stop_flag(clusters_arr))
        {
          propagateSplitterTagsKernel <<< dimGrid1, dimBlock1>>>(cell_state_arr, temporaries, clusters_arr, cell_info_arr,
                                                                 neighbour_pairs, pairs_number, share_cells);
          handleSplitterIndexChangesKernel <<< dimGrid2, dimBlock2>>>(cell_state_arr, temporaries, clusters_arr, cell_info_arr, neighbour_pairs, share_cells);
          handleSplitterTagChangesKernel <<< dimGrid3, dimBlock3>>>(cell_state_arr, temporaries, clusters_arr, cell_info_arr, neighbour_pairs, share_cells);
          checkForTagPropagationTermination <<< 1, 1>>>(cell_state_arr, temporaries, clusters_arr, cell_info_arr, neighbour_pairs, share_cells);
          //++counter;
        }
      //printf("COUNTS: %16d\n", counter);
#endif
    }
}

//run the kernel
void splitClusterGrowing(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temps,
                         const ConstantDataHolder & instance_data, const TASOptionsHolder & options, const bool synchronize,
                         CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  hipMemsetAsync(TASHacks::get_continue_flag_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
  hipMemsetAsync(TASHacks::get_reset_counter_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
#if !CAN_USE_TAIL_LAUNCH
  hipMemsetAsync(TASHacks::get_stop_flag_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
#endif
  splitClusterTagPropagationKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_cell_state_dev, temps, holder.m_clusters_dev,
                                                                 holder.m_cell_info_dev, holder.m_pairs_dev, options.m_options_dev);
  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}

/******************************************************************************************
 * Calculate the cell weights (only if indeed using shared_cells).
 ******************************************************************************************/

namespace
{
  constexpr int clusterprop_abs_E = 0;
  constexpr int clusterprop_E = 1;
  constexpr int clusterprop_x = 2;
  constexpr int clusterprop_y = 3;
  constexpr int clusterprop_z = 4;
}


__global__ static
void sumCellsForCentroid( Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                          Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                          const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                          const Helpers::CUDA_kernel_object<GeometryArr> geometry)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const TASTag tag = cell_state_arr->clusterTag[index];
      if (tag.is_part_of_splitter_cluster() && !tag.is_shared())
        {
          const int cluster = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, tag.index()) & 0xFFFF;

          const float energy = cell_info_arr->energy[index];
          const float abs_energy = fabsf(energy);
          const float x = geometry->x[index];
          const float y = geometry->y[index];
          const float z = geometry->z[index];

          atomicAdd( &( temporaries->get_cluster_property_aux_array<clusterprop_abs_E>(cluster) ), abs_energy    );
          atomicAdd( &( temporaries->get_cluster_property_aux_array<clusterprop_E>(cluster) ), energy        );

          atomicAdd( &( temporaries->get_cluster_property_aux_array<clusterprop_x>(cluster) ), x * abs_energy );
          atomicAdd( &( temporaries->get_cluster_property_aux_array<clusterprop_y>(cluster) ), y * abs_energy );
          atomicAdd( &( temporaries->get_cluster_property_aux_array<clusterprop_z>(cluster) ), z * abs_energy );

        }
    }
}

__global__ static
void calculateCentroids(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                        Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                        const int cluster_number)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < cluster_number)
    {

      const float abs_energy = temporaries->get_cluster_property_aux_array<clusterprop_abs_E>(i);

      if (abs_energy > 0)
        {
          temporaries->get_cluster_property_aux_array<clusterprop_x>(i) /= abs_energy; // x

          temporaries->get_cluster_property_aux_array<clusterprop_y>(i) /= abs_energy; // y

          temporaries->get_cluster_property_aux_array<clusterprop_z>(i) /= abs_energy; // z
        }
    }
}

__global__ static
void calculateCentroidsDeferKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                   Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int i_dimBlock1 = CalculateCentroidsBlockSize;
      const int i_dimGrid1 = Helpers::int_ceil_div(NCaloCells, i_dimBlock1);
      const dim3 dimBlock1(i_dimBlock1, 1, 1);
      const dim3 dimGrid1(i_dimGrid1, 1, 1);
#if CAN_USE_TAIL_LAUNCH
      calculateCentroids <<< dimGrid1, dimBlock1, 0, cudaStreamTailLaunch>>>(clusters_arr, temporaries, clusters_arr->number);
#else
      calculateCentroids <<< dimGrid1, dimBlock1>>>(clusters_arr, temporaries, clusters_arr->number);
#endif
    }
}


__global__ static
void assignFinalCells( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                       Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                       const Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                       const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                       const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const TASTag tag = cell_state_arr->clusterTag[index];
      if (tag.is_part_of_splitter_cluster())
        {
          if (opts->share_border_cells && tag.is_shared())
            {
              const uint32_t shared_clusters_packed = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, tag.index());
              const int cluster_1 = shared_clusters_packed & 0xFFFFU;
              const int cluster_2 = (shared_clusters_packed >> 16) & 0xFFFFU;

              const float cell_x = geometry->x[index];
              const float cell_y = geometry->y[index];
              const float cell_z = geometry->z[index];


              const float delta_x_1 = cell_x - temporaries->get_cluster_property_aux_array<clusterprop_x>(cluster_1);
              const float delta_x_2 = cell_x - temporaries->get_cluster_property_aux_array<clusterprop_x>(cluster_2);

              const float delta_y_1 = cell_y - temporaries->get_cluster_property_aux_array<clusterprop_y>(cluster_1);
              const float delta_y_2 = cell_y - temporaries->get_cluster_property_aux_array<clusterprop_y>(cluster_2);

              const float delta_z_1 = cell_z - temporaries->get_cluster_property_aux_array<clusterprop_z>(cluster_1);
              const float delta_z_2 = cell_z - temporaries->get_cluster_property_aux_array<clusterprop_z>(cluster_2);


              const float d_1 = sqrtf(delta_x_1 * delta_x_1 + delta_y_1 * delta_y_1 + delta_z_1 * delta_z_1);

              const float d_2 = sqrtf(delta_x_2 * delta_x_2 + delta_y_2 * delta_y_2 + delta_z_2 * delta_z_2);

              float r_exp = (d_1 - d_2) / opts->EM_shower_scale;

              if (r_exp > 10)
                {
                  r_exp = 10;
                }
              else if (r_exp < -10)
                {
                  r_exp = -10;
                }

              const float r = expf(r_exp);
              const float r_reverse = expf(-r_exp);

              float E_1 = temporaries->get_cluster_property_aux_array<clusterprop_E>(cluster_1);

              float E_2 = temporaries->get_cluster_property_aux_array<clusterprop_E>(cluster_2);

              if (opts->use_absolute_energy)
                {
                  E_1 = fabsf(E_1);
                  E_2 = fabsf(E_2);
                }

              if (E_1 <= 0)
                {
                  E_1 = 1.0f * CLHEP::MeV;
                }
              if (E_2 <= 0)
                {
                  E_2 = 1.0f * CLHEP::MeV;
                }

              float weight = E_1 / (E_1 + E_2 * r);
              float rev_weight = E_2 / (E_2 + E_1 * r_reverse);

              if (__float_as_uint(weight) == 0)
                {
                  weight == __uint_as_float(1);
                }

              if (__float_as_uint(rev_weight) == 0)
                {
                  rev_weight == __uint_as_float(1);
                }

              //This is just so that shared clusters
              //always show up as shared clusters.
              //A denormal weight is... negligible for physics.

              if (weight > 0.5f)
                {
                  cell_state_arr->clusterTag[index] = ClusterTag::make_tag(cluster_1, __float_as_uint(rev_weight), cluster_2);
                }
              else if (weight == 0.5f)
                {
                  const int max_cluster = cluster_1 > cluster_2 ? cluster_1 : cluster_2;
                  const int min_cluster = cluster_1 > cluster_2 ? cluster_2 : cluster_1;
                  cell_state_arr->clusterTag[index] = ClusterTag::make_tag(max_cluster, __float_as_uint(weight), min_cluster);
                }
              else /*if (weight < 0.5f)*/
                {
                  cell_state_arr->clusterTag[index] = ClusterTag::make_tag(cluster_2, __float_as_uint(weight), cluster_1);
                }
            }
          else
            {
              const int this_cluster = TASHacks::get_cell_to_cluster_table_entry(clusters_arr, tag.index()) & 0xFFFF;
              cell_state_arr->clusterTag[index] = ClusterTag::make_tag(this_cluster);
            }
        }
      else if (tag.is_non_assigned_part_of_split_cluster())
        {
          const int this_cluster = tag.index();
          cell_state_arr->clusterTag[index] = ClusterTag::make_tag(this_cluster);
          //Cells that are part of a pre-splitter cluster get added to the "same"?

          atomicMax(&(clusters_arr->seedCellID[this_cluster]), index);
          //Not the seed cell, but just a consistent way of marking this cluster as still valid...
        }
      else if (tag.is_part_of_original_cluster())
        {
          const int this_cluster = tag.index();
          cell_state_arr->clusterTag[index] = ClusterTag::make_tag(this_cluster);
        }
      else
        {
          cell_state_arr->clusterTag[index] = ClusterTag::make_invalid_tag();
        }
    }
}

//run the kernel
void cellWeightingAndFinalization(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temps,
                                  const ConstantDataHolder & instance_data, const TASOptionsHolder & options, const bool synchronize,
                                  CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  if (options.m_options->share_border_cells)
    {
      hipMemsetAsync(temps->secondaryArray, 0, sizeof(tag_type) * NCaloCells, hipStreamPerThread);
      const int i_dimBlock1 = SumCellsBlockSize;
      const int i_dimGrid1 = Helpers::int_ceil_div(NCaloCells, i_dimBlock1);
      const dim3 dimBlock1(i_dimBlock1, 1, 1);
      const dim3 dimGrid1(i_dimGrid1, 1, 1);

      sumCellsForCentroid <<< dimGrid1, dimBlock1, 0, stream_to_use>>>( holder.m_clusters_dev, temps,
                                                                        holder.m_cell_state_dev, holder.m_cell_info_dev,
                                                                        instance_data.m_geometry_dev                       );

      calculateCentroidsDeferKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_clusters_dev, temps);

    }

  const int i_dimBlock2 = FinalizeWeightsBlockSize;
  const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells, i_dimBlock2);
  const dim3 dimBlock2(i_dimBlock2, 1, 1);
  const dim3 dimGrid2(i_dimGrid2, 1, 1);

  assignFinalCells <<< dimGrid2, dimBlock2, 0, stream_to_use>>>( holder.m_cell_state_dev, holder.m_clusters_dev, temps,
                                                                 instance_data.m_geometry_dev, options.m_options_dev     );

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}
