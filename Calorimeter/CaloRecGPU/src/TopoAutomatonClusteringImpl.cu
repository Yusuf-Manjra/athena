#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "TopoAutomatonClusteringImpl.h"


#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include <cstddef>

using namespace CaloRecGPU;

void TACOptionsHolder::allocate()
{
  m_options.allocate();
  m_options_dev.allocate();
}

void TACOptionsHolder::sendToGPU(const bool clear_CPU)
{
  m_options_dev = m_options;
  if (clear_CPU)
    {
      m_options.clear();
    }
}

//constexpr static int DefaultBlockSize = 256;

constexpr static int SignalToNoiseBlockSize = 512;
constexpr static int CellPairsBlockSize = 256;
constexpr static int ClusterGrowingPropagationBlockSize = 256;
constexpr static int ClusterGrowingCopyAndCheckBlockSize = 512;
constexpr static int ClusterGrowingTerminalPropagationBlockSize = 256;
constexpr static int ClusterGrowingFinalizationBlockSize = 512;
constexpr static int ClusterGrowingSeedCellAssignmentBlockSize = 256;

#if defined(__CUDA_ARCH__) &&  __CUDA_ARCH__ > 350
  #if CUDART_VERSION >= 12000
    #define CAN_USE_TAIL_LAUNCH 1
  #else
    #define CAN_USE_TAIL_LAUNCH 0
  #endif
#elif defined(__CUDA_ARCH__)
  #error "CUDA compute capability at least 3.5 is needed so we can have dynamic parallelism!"
#endif

namespace TACHacks
//We will (ab)use the cluster info to hold
//the cell-to-cluster map and the continue flag,
//as the cluster properties aren't set here.
//Some pointer trickery, but, given that CUDA allocations
//are, in their essence, casted from void * as in C,
//this should be safe, even more since int and float are both 32-bit.
{
  //We must do this this way because there are more cells than NMaxClusters,
  //so we spill over to the next cluster properties (Phi and the first ~third of seedCellID)
  static __host__ __device__ int * get_cell_to_cluster_table_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                                     const int index)
  {
    //void * v_ptr = &(clusters_arr->clusterEt[0]);
    void * v_ptr = &(clusters_arr->clusterEta[4]);
    //The first 3 elements will possibly be taken over by
    //the seed array, and an offset of 4 feels more "natural"...
    return ((int *) v_ptr) + index;
  }

  static __device__ int get_cell_to_cluster_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                        const int index)
  {
    return (*get_cell_to_cluster_table_address(clusters_arr, index));
  }

  static __device__ void set_cell_to_cluster_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                         const int index, const int new_value)
  {
    (*get_cell_to_cluster_table_address(clusters_arr, index)) = new_value;
  }

  static __device__ int get_continue_flag(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return clusters_arr->seedCellID[NMaxClusters - 1];
  }

  static __device__ void set_continue_flag(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int value)
  {
    clusters_arr->seedCellID[NMaxClusters - 1] = value;
  }


  static __host__ __device__ int * get_continue_flag_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return &(clusters_arr->seedCellID[NMaxClusters - 1]);
  }


  static __host__ __device__ unsigned long long int * get_seed_cell_table_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                                                  const int index)
  {
    void * base_ptr = (ClusterInfoArr *) clusters_arr;
    return ((unsigned long long int *) base_ptr ) + index + 1;
    //We have the numbers first...
    //Of course the ClusterInfoArr
    //will be 8-byte aligned,
    //thus any 8-byte offset from it will also be.
  }

  static __device__ unsigned long long int get_seed_cell_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                                     const int index)
  {
    return (*get_seed_cell_table_address(clusters_arr, index));
  }

  static __device__ void set_seed_cell_table_entry(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                   const int index, const unsigned long long int new_value)
  {
    (*get_seed_cell_table_address(clusters_arr, index)) = new_value;
  }

#if !CAN_USE_TAIL_LAUNCH

  static __device__ int get_stop_flag(const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return clusters_arr->seedCellID[NMaxClusters - 2];
  }

  static __device__ void set_stop_flag(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int value)
  {
    clusters_arr->seedCellID[NMaxClusters - 2] = value;
  }

  static __host__ __device__ int * get_stop_flag_address(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
  {
    return &(clusters_arr->seedCellID[NMaxClusters - 2]);
  }

#endif
}

/******************************************************************************
 * Kernel to compute the cells signal (Energy) to noise ratio, used for the
 * cells clustering step, to define the seed cells, the growing cells and the
 * border cells.
 * It also apply the energy thresholds (seed-4, grow-2, border-0)
 ******************************************************************************/

static __global__
void signalToNoiseKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                          Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                          const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                          const Helpers::CUDA_kernel_object<CellNoiseArr> noise_arr,
                          const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                          const Helpers::CUDA_kernel_object<TopoAutomatonOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;


  if (index < NCaloCells)
    {
      const int cell_sampling = geometry->caloSample[index];
      const float cellEnergy = cell_info_arr->energy[index];

      if (!cell_info_arr->is_valid(index) || !opts->uses_calorimeter_by_sampling(cell_sampling))
        {
          cell_state_arr->clusterTag[index] = TACTag::make_invalid_tag();
          temporaries->secondaryArray[index] = TACTag::make_invalid_tag();
          return;
        }

      float sigNoiseRatio = 0.00001f;
      //It's what's done in the CPU implementation...
      if (!cell_info_arr->is_bad(*geometry, index, opts->treat_L1_predicted_as_good))
        {
          const int gain = cell_info_arr->gain[index];

          float cellNoise = 0.f;
          if (opts->use_two_gaussian && geometry->is_tile(index))
            {
              //getTileEffSigma
            }
          else
            {
              cellNoise = noise_arr->noise[gain][index];
            }
          if (isfinite(cellNoise) && cellNoise > 0.0f)
            {
              sigNoiseRatio = cellEnergy / cellNoise;
            }
        }

      const float absRatio = fabsf(sigNoiseRatio);

      bool can_be_seed = (opts->abs_seed ? absRatio : sigNoiseRatio) > opts->seed_threshold;
      bool can_be_grow = (opts->abs_grow ? absRatio : sigNoiseRatio) > opts->grow_threshold;
      bool can_be_term = (opts->abs_terminal ? absRatio : sigNoiseRatio) > opts->terminal_threshold;

      if (can_be_seed && opts->use_time_cut && (!opts->keep_significant_cells || sigNoiseRatio <= opts->snr_threshold_for_keeping_cells))
        {
          if (!cell_info_arr->passes_time_cut(*geometry, index, opts->time_threshold))
            {
              can_be_seed = false;
              if (opts->completely_exclude_cut_seeds)
                {
                  can_be_grow = false;
                  can_be_term = false;
                }
            }
        }


      if (can_be_seed && opts->uses_seed_sampling(cell_sampling))
        {
          const int n = atomicAdd(&(clusters_arr->number), 1);

          const TACTag tag = TACTag::make_seed_tag(index, __float_as_int(absRatio), can_be_grow);
          //Since seed_threshold will be positive,
          //no problem with using abs here always:
          //when actually using the absolute value,
          //it's what we want, when not, cells with
          //negative SNR will not be acceptable seeds.
          //
          //As per the CPU algorithm,
          //if a cell does not pass the grow threshold
          //(which can happen if seeds are being evaluated
          // as absolute value while growing cells are not),
          //the clusters cannot be merged. Somehow.

          cell_state_arr->clusterTag[index] = tag;
          temporaries->secondaryArray[index] = tag;

          //TACHacks::set_merge_table_entry(clusters_arr, n, n);
          TACHacks::set_cell_to_cluster_table_entry(clusters_arr, index, n);

          unsigned long long int snr_and_cell = __float_as_uint(absRatio);

          snr_and_cell = (snr_and_cell << 32) | index;

          TACHacks::set_seed_cell_table_entry(clusters_arr, n, snr_and_cell);
        }
      else if (can_be_grow)
        {
          cell_state_arr->clusterTag[index] = TACTag::make_grow_tag();
          temporaries->secondaryArray[index] = TACTag::make_grow_tag();
        }
      else if (can_be_term)
        {
          cell_state_arr->clusterTag[index] = TACTag::make_terminal_tag();
          temporaries->secondaryArray[index] = TACTag::make_terminal_tag();
        }
      else //is invalid for propagation
        {
          cell_state_arr->clusterTag[index] = TACTag::make_invalid_tag();
          temporaries->secondaryArray[index] = TACTag::make_invalid_tag();
        }
    }
}

//run the kernel
void signalToNoise(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temps,
                   const ConstantDataHolder & instance_data, const TACOptionsHolder & options, const bool synchronize,
                   CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  hipMemsetAsync(&(holder.m_clusters_dev->number), 0, sizeof(holder.m_clusters_dev->number), stream_to_use);

  const int i_dimBlock = SignalToNoiseBlockSize;
  const int i_dimGrid = Helpers::int_ceil_div(NCaloCells, i_dimBlock);

  const dim3 dimBlock(i_dimBlock, 1, 1);
  const dim3 dimGrid(i_dimGrid, 1, 1);
  signalToNoiseKernel <<< dimGrid, dimBlock, 0, stream_to_use>>>(holder.m_cell_state_dev, holder.m_clusters_dev, temps,
                                                                 holder.m_cell_info_dev, instance_data.m_cell_noise_dev, instance_data.m_geometry_dev,
                                                                 options.m_options_dev);
  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/******************************************************************************
 * Kernel to generate the cell pairs for the growing algorithm.
 ******************************************************************************/


static __global__
void cellPairsKernel( Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                      const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                      const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                      const Helpers::CUDA_kernel_object<TopoAutomatonOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < NCaloCells)
    {
      const TACTag this_tag = cell_state_arr->clusterTag[index];

      if (this_tag.is_grow_or_seed())
        {
          int full_neighs[NMaxNeighbours], grow_neighs[NMaxNeighbours], term_neighs[NMaxNeighbours];

          int num_grow_neighs = 0, num_term_neighs = 0;

          const int num_neighs = geometry->neighbours.get_neighbours_with_option(opts->neighbour_options, index, full_neighs,
                                                                                 opts->limit_HECIW_and_FCal_neighs, opts->limit_PS_neighs);

          for (int i = 0; i < num_neighs; ++i)
            {
              const int neigh_ID = full_neighs[i];
              const TACTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];
              if (neigh_tag.is_grow_or_seed())
                {
                  grow_neighs[num_grow_neighs] = neigh_ID;
                  ++num_grow_neighs;
                }
              else if (neigh_tag.is_non_assigned_terminal())
                {
                  term_neighs[num_term_neighs] = neigh_ID;
                  ++num_term_neighs;
                }
            }

          if (num_grow_neighs > 0)
            {
              const int n = atomicAdd(&(neighbour_pairs->number), num_grow_neighs);
              for (int i = 0; i < num_grow_neighs; ++i)
                {
                  neighbour_pairs->cellID[n + i] = grow_neighs[i];
                  neighbour_pairs->neighbourID[n + i] = index;
                }
            }
          if (num_term_neighs > 0)
            {
              const int n = atomicAdd(&(neighbour_pairs->reverse_number), num_term_neighs);
              const int real_start = NMaxPairs - n - num_term_neighs;
              for (int i = 0; i < num_term_neighs; ++i)
                {
                  neighbour_pairs->cellID[real_start + i] = term_neighs[i];
                  neighbour_pairs->neighbourID[real_start + i] = index;
                }
            }
        }
    }
}

//run the kernel
void cellPairs(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> /*temps*/,
               const ConstantDataHolder & instance_data, const TACOptionsHolder & options, const bool synchronize,
               CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  hipMemsetAsync(&(holder.m_pairs_dev->number), 0, sizeof(holder.m_pairs_dev->number), stream_to_use);
  hipMemsetAsync(&(holder.m_pairs_dev->reverse_number), 0, sizeof(holder.m_pairs_dev->reverse_number), stream_to_use);

  const int i_dimBlock = CellPairsBlockSize;
  const int i_dimGrid = Helpers::int_ceil_div(NCaloCells, i_dimBlock);

  const dim3 dimBlock(i_dimBlock, 1, 1);
  const dim3 dimGrid(i_dimGrid, 1, 1);

  cellPairsKernel <<< dimGrid, dimBlock, 0, stream_to_use>>>(holder.m_pairs_dev, holder.m_cell_state_dev, instance_data.m_geometry_dev, options.m_options_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}



/******************************************************************************
 * Series of kernels for the growing algorithm!
 ******************************************************************************/
__global__ static
void propagateNeighbours( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                          Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                          const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                          const int pair_number);

__global__ static
void propagateTerminals( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                         const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                         const int reverse_pair_number);

__global__ static
void copyTagsAndCheckTermination( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                  Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                                  Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                  const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs         );

__global__ static
void finalizeClusterAttribution(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                const Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                                Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr);

__global__ static
void assignSeedCells(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int clusters_number);

__global__ static
void propagateNeighbours( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                          Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                          const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                          const int pair_number)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < pair_number)
    {
      const int this_ID = neighbour_pairs->cellID[index];
      const int neigh_ID = neighbour_pairs->neighbourID[index];

      const TACTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];

      const TACTag prop_tag = neigh_tag.propagate();

      const TACTag this_old_tag = cell_state_arr->clusterTag[this_ID];

      if (this_old_tag.is_part_of_cluster() && neigh_tag.is_part_of_cluster() && this_old_tag.can_merge())
        {
          //If the cell was already part of a cluster,
          //we must merge the two of them.
          //Else, we keep growing.
          const int this_seed_idx = this_old_tag.index();
          const int neigh_seed_idx = neigh_tag.index();
          if (this_seed_idx != neigh_seed_idx)
            {
              const int this_cluster_index = TACHacks::get_cell_to_cluster_table_entry(clusters_arr, this_seed_idx);
              const int neigh_cluster_index = TACHacks::get_cell_to_cluster_table_entry(clusters_arr, neigh_seed_idx);
              if (this_cluster_index != neigh_cluster_index)
                {
                  TACHacks::set_continue_flag(clusters_arr, 1);
                  if (this_cluster_index > neigh_cluster_index)
                    {
                      atomicMax(TACHacks::get_cell_to_cluster_table_address(clusters_arr, neigh_seed_idx), this_cluster_index);
                      atomicMax( TACHacks::get_seed_cell_table_address(clusters_arr, this_cluster_index),
                                 TACHacks::get_seed_cell_table_entry(clusters_arr, neigh_cluster_index)    );
                    }
                  else /* if (neigh_cluster_index > this_cluster_index) */
                    {
                      atomicMax(TACHacks::get_cell_to_cluster_table_address(clusters_arr, this_seed_idx), neigh_cluster_index);
                      atomicMax( TACHacks::get_seed_cell_table_address(clusters_arr, neigh_cluster_index),
                                 TACHacks::get_seed_cell_table_entry(clusters_arr, this_cluster_index)    );
                    }
                }
            }
          if (prop_tag > this_old_tag)
            {
              atomicMax(&(temporaries->secondaryArray[this_ID]), prop_tag);
              TACHacks::set_continue_flag(clusters_arr, 1);
            }
        }
      else if (!this_old_tag.is_part_of_cluster() && neigh_tag.is_part_of_cluster())
        {
          TACHacks::set_continue_flag(clusters_arr, 1);
          atomicMax(&(temporaries->secondaryArray[this_ID]), prop_tag);
        }
    }
#if CAN_USE_TAIL_LAUNCH
  else if (index == pair_number)
    {
      const int i_dimBlock = ClusterGrowingCopyAndCheckBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(NCaloCells + 1, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);

      copyTagsAndCheckTermination <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs);

    }

#endif
}


__global__ static
void copyTagsAndCheckTermination( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                  Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                                  Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                  const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs  )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      cell_state_arr->clusterTag[index] = temporaries->secondaryArray[index];
    }
  else if (index == NCaloCells)
    {
#if CAN_USE_TAIL_LAUNCH
      if (TACHacks::get_continue_flag(clusters_arr))
        {
          const int pairs_number = neighbour_pairs->number;

          const int i_dimBlock = ClusterGrowingPropagationBlockSize;
          const int i_dimGrid = Helpers::int_ceil_div(pairs_number + 1, i_dimBlock);
          const dim3 dimBlock(i_dimBlock, 1, 1);
          const dim3 dimGrid(i_dimGrid, 1, 1);

          TACHacks::set_continue_flag(clusters_arr, 0);

          propagateNeighbours <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs, pairs_number);
        }
      else
        {

          const int reverse_pairs_number = neighbour_pairs->reverse_number;
          const int i_dimBlock = ClusterGrowingTerminalPropagationBlockSize;
          const int i_dimGrid = Helpers::int_ceil_div(reverse_pairs_number + 1, i_dimBlock);
          const dim3 dimBlock(i_dimBlock, 1, 1);
          const dim3 dimGrid(i_dimGrid, 1, 1);
          propagateTerminals <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs, reverse_pairs_number);

        }
#else
      if (!TACHacks::get_continue_flag(clusters_arr))
        {
          TACHacks::set_stop_flag(clusters_arr, 1);
        }
      else
        {
          TACHacks::set_continue_flag(clusters_arr, 0);
        }
#endif
    }

}

__global__ static
void propagateTerminals( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                         const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs,
                         const int reverse_pair_number)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < reverse_pair_number)
    {
      const int start_index = NMaxPairs - reverse_pair_number;
      const int this_ID = neighbour_pairs->cellID[start_index + index];
      const int neigh_ID = neighbour_pairs->neighbourID[start_index + index];

      const TACTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];

      atomicMax(&(temporaries->secondaryArray[this_ID]), neigh_tag.propagate());
    }
#if CAN_USE_TAIL_LAUNCH
  else if (index == reverse_pair_number)
    {
      const int i_dimBlock = ClusterGrowingFinalizationBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(NCaloCells + 1, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);

      finalizeClusterAttribution <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr);
    }
#endif
}

__global__ static
void finalizeClusterAttribution(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                const Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                                Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < NCaloCells)
    {
      const TACTag old_tag = temporaries->secondaryArray[index];

      if (old_tag.is_part_of_cluster())
        {
          cell_state_arr->clusterTag[index] = ClusterTag::make_tag(TACHacks::get_cell_to_cluster_table_entry(clusters_arr, old_tag.index()));
        }
      else
        {
          cell_state_arr->clusterTag[index] = ClusterTag::make_invalid_tag();
        }
    }
#if CAN_USE_TAIL_LAUNCH
  else if (index == NCaloCells)
    {
      const int num_clusters = clusters_arr->number;
      const int i_dimBlock = ClusterGrowingSeedCellAssignmentBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(clusters_arr->number, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);

      assignSeedCells <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(clusters_arr, num_clusters);
    }
#endif
}

__global__ static
void assignSeedCells(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr, const int clusters_number)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < clusters_number)
    {
      const unsigned long long int SNR_and_cell = TACHacks::get_seed_cell_table_entry(clusters_arr, index);
      const int cell = SNR_and_cell & 0xFFFFFU;
      clusters_arr->seedCellID[index] = cell;
      //This was built in such a way there's no overlap between the parts we access...
    }
}

__global__ static
void clusterGrowingKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                           Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temporaries,
                           Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                           const Helpers::CUDA_kernel_object<PairsArr> neighbour_pairs)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int pairs_number = neighbour_pairs->number;

      const int i_dimBlock1 = ClusterGrowingPropagationBlockSize;
      const int i_dimGrid1 = Helpers::int_ceil_div(pairs_number + 1, i_dimBlock1);
      const dim3 dimBlock1(i_dimBlock1, 1, 1);
      const dim3 dimGrid1(i_dimGrid1, 1, 1);

#if CAN_USE_TAIL_LAUNCH
      propagateNeighbours <<< dimGrid1, dimBlock1, 0, cudaStreamTailLaunch>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs, pairs_number);
#else

      const int i_dimBlock2 = ClusterGrowingCopyAndCheckBlockSize;
      const int i_dimGrid2 = Helpers::int_ceil_div(NCaloCells + 1, i_dimBlock2);
      const dim3 dimBlock2(i_dimBlock2, 1, 1);
      const dim3 dimGrid2(i_dimGrid2, 1, 1);

      const int reverse_pairs_number = neighbour_pairs->reverse_number;

      const int i_dimBlock3 = ClusterGrowingTerminalPropagationBlockSize;
      const int i_dimGrid3 = Helpers::int_ceil_div(reverse_pairs_number, i_dimBlock3);
      const dim3 dimBlock3(i_dimBlock3, 1, 1);
      const dim3 dimGrid3(i_dimGrid3, 1, 1);

      const int i_dimBlock4 = ClusterGrowingFinalizationBlockSize;
      const int i_dimGrid4 = Helpers::int_ceil_div(NCaloCells, i_dimBlock4);
      const dim3 dimBlock4(i_dimBlock4, 1, 1);
      const dim3 dimGrid4(i_dimGrid4, 1, 1);

      const int i_dimBlock5 = ClusterGrowingSeedCellAssignmentBlockSize;
      const int i_dimGrid5 = Helpers::int_ceil_div(clusters_arr->number, i_dimBlock5);
      const dim3 dimBlock5(i_dimBlock5, 1, 1);
      const dim3 dimGrid5(i_dimGrid5, 1, 1);

      //int counter = 0;

      while (!TACHacks::get_stop_flag(clusters_arr))
        {
          propagateNeighbours <<< dimGrid1, dimBlock1>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs, pairs_number);
          copyTagsAndCheckTermination <<< dimGrid2, dimBlock2>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs);

          //++counter;

        }

      //printf("COUNTS: %16d\n", counter);

      propagateTerminals <<< dimGrid3, dimBlock3>>>(cell_state_arr, temporaries, clusters_arr, neighbour_pairs, reverse_pairs_number);
      finalizeClusterAttribution <<< dimGrid4, dimBlock4>>>(cell_state_arr, temporaries, clusters_arr);
      assignSeedCells <<< dimGrid5, dimBlock5>>>(clusters_arr, clusters_arr->number);
#endif
    }
}

//run the kernel
void clusterGrowing(EventDataHolder & holder, Helpers::CUDA_kernel_object<TopoAutomatonTemporaries> temps,
                    const ConstantDataHolder & instance_data, const TACOptionsHolder & options, const bool synchronize,
                    CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{

  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  hipMemsetAsync(TACHacks::get_continue_flag_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
#if !CAN_USE_TAIL_LAUNCH
  hipMemsetAsync(TACHacks::get_stop_flag_address(holder.m_clusters_dev), 0, sizeof(int), stream_to_use);
#endif
  clusterGrowingKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_cell_state_dev, temps, holder.m_clusters_dev, holder.m_pairs_dev);


  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}
