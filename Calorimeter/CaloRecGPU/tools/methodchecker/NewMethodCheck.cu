#include "hip/hip_runtime.h"
// Dear emacs, this is -*- c++ -*-
#define TTAC_CALCULATE_PHI_BY_SIMPLE_AVERAGE 1

#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <unordered_map>
#include <map>
#include <set>
#include <algorithm>
#include <numeric>

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "CaloRecGPU/StandaloneDataIO.h"

#include "../../src/TopoAutomatonClusteringGPU.cu"
#include "../../src/BasicGPUClusterInfoCalculatorImpl.cu"

#include <chrono>

using namespace CaloRecGPU;

void generic_comparison_print(const std::vector<size_t> & vect, const std::string & title, const size_t break_after)
{
  std::cout << "\n\n " << title << ":";

  size_t accum = 0, local_accum = 0;

  for (size_t i = 0; i < vect.size(); ++i)
    {
      if (i % break_after == 0)
        {
          if (i > 0)
            {
              std::cout << "(" << local_accum / double(break_after) << ")";
            }
          local_accum = 0;
          std::cout << "\n             ";
        }
      const size_t res = vect[i];
      printf("%5zu ", res);
      accum += res;
      local_accum += res;
    }
  std::cout << "(" << local_accum / double(break_after) << ")";
  std::cout << "\n (" << accum / double(vect.size()) << ")";
}

struct Results
{
  Helpers::CPU_object<CellStateArr> m_state;

  std::vector<size_t> total, assignment, terminal, inclusion;

  bool has_total = false, has_assignment = false, has_terminal = false, has_inclusion = false;

  void set_reference(const EventDataHolder & event_data)
  {
    m_state = event_data.m_cell_state_dev;
  }

  void add_comparison(const EventDataHolder & event_data)
  {
    Helpers::CPU_object<CellStateArr> temp_state = event_data.m_cell_state_dev;
    size_t total_diffs = 0, just_assignment = 0, terminal_weird = 0, inclusion_or_not = 0;
    for (int i = 0; i < NCaloCells; ++i)
      {
        const tag_type test_tag = temp_state->clusterTag[i];
        const tag_type ref_tag = m_state->clusterTag[i];
        if (test_tag != ref_tag)
          {
            ++total_diffs;
            has_total = true;
            if (Tags::is_part_of_cluster(test_tag) && Tags::is_part_of_cluster(ref_tag))
              {
                ++just_assignment;
                has_assignment = true;
              }
            else if (Tags::is_non_assigned_terminal(test_tag) && Tags::is_non_assigned_terminal(ref_tag))
              {
                ++terminal_weird;
                has_terminal = true;
              }
            else
              {
                ++inclusion_or_not;
                has_inclusion = true;
              }
          }
      }
    total.push_back(total_diffs);
    assignment.push_back(just_assignment);
    terminal.push_back(terminal_weird);
    inclusion.push_back(inclusion_or_not);
  }

  void print_comparison(const size_t break_after, const bool force_print)
  {
    std::cout << " --------- CONSISTENCY --------- \n\n";

    if (has_total || force_print)
      {
        generic_comparison_print(total, "Total", break_after);
      }

    if (has_assignment || force_print)
      {
        generic_comparison_print(assignment, "Assignment", break_after);
      }


    if (has_terminal || force_print)
      {
        generic_comparison_print(terminal, "Terminal", break_after);
      }

    if (has_inclusion || force_print)
      {
        generic_comparison_print(inclusion, "Inclusion", break_after);
      }

    if (!has_total)
      {
        std::cout << "\nAll good, you can go rest now.\n";
      }
    std::cout << std::endl;

  }

};


struct CPUComparison
{

  struct SNRArray
  {
    float snr[NCaloCells];
  };

  Helpers::CPU_object<CellStateArr> m_state;
  Helpers::CPU_object<SNRArray> m_snr_arr;

  int m_ref_clusters = 0;

  std::vector<size_t> total, seed, grow, term;

  bool has_total = false, has_seed = false, has_grow = false, has_term = false;

  struct phi_errors
  {
    double mu = 0, sigma_sqr = 0;
    double differences = 0;
    double max = std::numeric_limits<double>::lowest();
    void add (const double val)
    {
      mu += val;
      sigma_sqr += val * val;
      max = std::max(max, val);
    }
    void add_difference()
    {
      differences += 1;
    }
    void finalize(const size_t count, const size_t num_events = 1)
    {
      mu /= count;
      sigma_sqr /= count;
      sigma_sqr -= mu * mu;
      differences /= num_events;
    }
    void combine(const phi_errors & dp, const size_t count)
    {
      mu += dp.mu * count;
      sigma_sqr += (dp.sigma_sqr + dp.mu * dp.mu) * count;
      differences += dp.differences;
      max = std::max(max, dp.max);
    }

    template <class stream>
    friend stream & operator<< (stream & s, const phi_errors & pe)
    {
      s << "|" << pe.differences << " " << pe.max << " " << pe.mu + (pe.mu < 1e-3) * 1e-3 << " " << std::sqrt(pe.sigma_sqr) << "|";
      return s;
    }

  };

  std::vector<std::string> phi_comparison_labels { "Reference to CPU-Calculated",
    "Reference to GPU-Calculated",
    "CPU-Calculated to GPU-Calculated",
    "CPU to CPU No Reg.",
    "CPU to CPU KBN",
    "CPU to CPU KBN No Reg.",
    "CPU to CPU Sin/Cos",
    "CPU to CPU Sin/Cos KBN",
    "CPU KBN No Reg. to GPU",
    "Reference No Reg. to GPU",
    "Reference Sin/Cos to GPU"};

  std::vector<std::vector<phi_errors>> phi_comparisons = std::vector<std::vector<phi_errors>>(phi_comparison_labels.size());

  std::vector<size_t> valid_phis_counts;

  template <class T>
  static inline T regularize(T b, T a)
  //a. k. a. proxim in Athena code.
  {
    const T aplus = a + Helpers::Constants::pi<T>;
    const T aminus = a - Helpers::Constants::pi<T>;
    if (b > aplus)
      {
        do
          {
            b -= 2 * Helpers::Constants::pi<T>;
          }
        while (b > aplus);
      }
    else if (b < aminus)
      {
        do
          {
            b += 2 * Helpers::Constants::pi<T>;
          }
        while (b < aminus);
      }
    return b;
  }
  static double angle_fix(const double angle)
  {
    return regularize(angle, 0.);
    //fmod, no?
  }

  struct base_phi_calc
  {
    virtual void resize(const size_t) = 0;
    virtual void update_cluster(const int cluster, const double energy, const double phi, const double seed_phi) = 0;
    virtual void finalize(std::vector<double> & array) = 0;
  };


  inline static
  void calculate_phis(base_phi_calc & phi_calculator, std::vector<double> & cluster_phi_array, const CellStateArr & state,
                      const int n_clusters, const float * phi, const float * snr, const float * energy)
  {

    std::vector<int> seed_cell_id(n_clusters, -1);
    std::vector<float> seed_snr(n_clusters, 0);

    for (int i = 0; i < NCaloCells; ++i)
      {
        const tag_type tag = state.clusterTag[i];
        if (Tags::is_part_of_cluster(tag))
          {
            const int index = Tags::get_index_from_tag(tag);
            const float this_snr = std::abs(snr[i]);
            if (this_snr > seed_snr[index])
              {
                seed_cell_id[index] = i;
                seed_snr[index] = this_snr;
              }
          }
      }

    std::vector<float> seed_phi(n_clusters, 0);

    for (int i = 0; i < n_clusters; ++i)
      {
        const int id = seed_cell_id[i];
        if (id < 0)
          {
            std::cout << "ERRRR! " << i << " " << id << std::endl;
          }
        else
          {
            seed_phi[i] = phi[id];
          }
      }

    phi_calculator.resize(n_clusters);

    for (int i = 0; i < NCaloCells; ++i)
      {
        const tag_type tag = state.clusterTag[i];
        if (Tags::is_part_of_cluster(tag))
          {
            const int index = Tags::get_index_from_tag(tag);
            phi_calculator.update_cluster(index, energy[i], phi[i], seed_phi[index]);
          }
      }

    cluster_phi_array.clear();
    cluster_phi_array.resize(n_clusters, -1000.);

    phi_calculator.finalize(cluster_phi_array);

  }

  struct standard_phi_calc : public base_phi_calc
  {
    std::vector<double> phis, weights;
    void resize(const size_t n_clusters)
    {
      phis.clear();
      phis.resize(n_clusters, 0);
      weights.clear();
      weights.resize(n_clusters, 0);
    }
    void update_cluster(const int cluster, const double energy, const double phi, const double seed_phi)
    {
      const double abs_e = std::abs(energy);
      weights[cluster] += abs_e;
      phis[cluster] += abs_e * regularize(phi, seed_phi);
    }
    void finalize(std::vector<double> & array)
    {
      for (size_t i = 0; i < phis.size(); ++i)
        {
          array[i] = angle_fix(phis[i] / weights[i]);
        }
    }
  };

  struct standard_phi_calc_no_reg : public standard_phi_calc
  {
    void update_cluster(const int cluster, const double energy, const double phi, const double seed_phi)
    {
      const double abs_e = std::abs(energy);
      weights[cluster] += abs_e;
      phis[cluster] += abs_e * phi;
    }
  };

  struct KBN_base_class : public base_phi_calc
  {
    template <class T> inline static
    void KahanBabushkaNeumaier_summate(const T res, T & Sum, T & Correct)
    {
      const T t = Sum + res;
      if (std::abs(Sum) >= std::abs(res))
        {
          Correct += (Sum - t) + res;
        }
      else
        {
          Correct += (res - t) + Sum;
        }
      Sum = t;
    }
  };

  struct KBN_phi_calc : public KBN_base_class
//Kahan-Babushka-Neumaier Summation
  {
    std::vector<double> phis, weights, corrections;
    void resize(const size_t n_clusters)
    {
      phis.clear();
      phis.resize(n_clusters, 0);
      weights.clear();
      weights.resize(n_clusters, 0);
      corrections.clear();
      corrections.resize(n_clusters, 0);
    }
    void update_cluster(const int cluster, const double energy, const double phi, const double seed_phi)
    {
      const double abs_e = std::abs(energy);
      weights[cluster] += abs_e;
      KahanBabushkaNeumaier_summate(abs_e * regularize(phi, seed_phi), phis[cluster], corrections[cluster]);
    }
    void finalize(std::vector<double> & array)
    {
      for (size_t i = 0; i < phis.size(); ++i)
        {
          array[i] = angle_fix((phis[i] + corrections[i]) / weights[i]);
        }
    }
  };

  struct KBN_phi_calc_no_reg : public KBN_phi_calc
  {
    void update_cluster(const int cluster, const double energy, const double phi, const double seed_phi)
    {
      const double abs_e = std::abs(energy);
      weights[cluster] += abs_e;
      KahanBabushkaNeumaier_summate(abs_e * phi, phis[cluster], corrections[cluster]);
    }
  };

  struct sincos_phi_calc : public base_phi_calc
  {
    std::vector<double> sins, coss;
    void resize(const size_t n_clusters)
    {
      sins.clear();
      sins.resize(n_clusters, 0);
      coss.clear();
      coss.resize(n_clusters, 0);
    }
    void update_cluster(const int cluster, const double energy, const double phi, const double seed_phi)
    {
      const double abs_e = std::abs(energy);
      sins[cluster] += abs_e * sin(phi);
      coss[cluster] += abs_e * cos(phi);
    }
    void finalize(std::vector<double> & array)
    {
      for (size_t i = 0; i < sins.size(); ++i)
        {
          const double raw_phi = atan2(sins[i], coss[i]);
          if (isnan(raw_phi) || isinf(raw_phi))
            {
              array[i] = -1000.;
            }
          else
            {
              array[i] = angle_fix(raw_phi);
            }
        }
    }
  };

  struct sincos_KBN_phi_calc : public KBN_base_class
  {
    std::vector<double> sins, coss, sincorr, coscorr;
    void resize(const size_t n_clusters)
    {
      sins.clear();
      sins.resize(n_clusters, 0);
      coss.clear();
      coss.resize(n_clusters, 0);
      sincorr.clear();
      sincorr.resize(n_clusters, 0);
      coscorr.clear();
      coscorr.resize(n_clusters, 0);
    }
    void update_cluster(const int cluster, const double energy, const double phi, const double seed_phi)
    {
      const double abs_e = std::abs(energy);
      KahanBabushkaNeumaier_summate(abs_e * sin(phi), sins[cluster], sincorr[cluster]);
      KahanBabushkaNeumaier_summate(abs_e * cos(phi), coss[cluster], coscorr[cluster]);
    }
    void finalize(std::vector<double> & array)
    {
      for (size_t i = 0; i < sins.size(); ++i)
        {
          const double raw_phi = atan2(sins[i] + sincorr[i], coss[i] + coscorr[i]);
          if (isnan(raw_phi) || isinf(raw_phi))
            {
              array[i] = -1000.;
            }
          else
            {
              array[i] = angle_fix(raw_phi);
            }
        }
    }
  };


  void set_reference(const CellStateArr & state, const CellInfoArr & cell_info, const ConstantDataHolder & instance_data, const int num_clusters)
  {
    m_ref_clusters = num_clusters;

    m_state.allocate();
    *(m_state) = state;

    m_snr_arr.allocate();

    for (int i = 0; i < NCaloCells; ++i)
      {
        float snr = 0.00001f;

        const int local_gain = cell_info.gain[i];

        if (GainConversion::is_invalid_cell(local_gain))
          {
            snr = 0;
            //I mean... These won't be used anyway, so... any value goes.
          }
        else if (GainConversion::is_normal_cell(local_gain) || GainConversion::is_invalid_seed_cell(local_gain))
          {
            const float local_noise = instance_data.m_cell_noise->noise[GainConversion::recover_invalid_seed_cell_gain(local_gain)][i];
            if (finite(local_noise) && local_noise > 0.0f)
              {
                snr = std::abs(cell_info.energy[i] / local_noise);
              }
          }
        m_snr_arr->snr[i] = snr;
      }

    std::vector<double> phi_calcs;
    standard_phi_calc spc{};
    calculate_phis(spc, phi_calcs, *m_state, m_ref_clusters,
                   instance_data.m_geometry->phi, m_snr_arr->snr, cell_info.energy);

  }

  static constexpr double default_min_similarity = 0.75;
  static constexpr double default_term_weight = 0.0;
  static constexpr double default_grow_weight = 250;
  static constexpr double default_seed_weight = 10000;


  static constexpr float SNR_thresholds[3] = {0., 2., 4.};

  void find_matches(std::vector<int> & r2t_matches,
                    std::vector<int> & t2r_matches,
                    const float * SNR_array,
                    const tag_type * ref_tag_array,
                    const tag_type * test_tag_array,
                    const size_t num_ref_clusters,
                    const size_t num_test_clusters,
                    const double min_similarity = default_min_similarity,
                    const double term_weight = default_term_weight,
                    const double grow_weight = default_grow_weight,
                    const double seed_weight = default_seed_weight)
  //Expects clusters numbered from 0 to N-1...
  {
    std::vector<int> similarity_map(num_ref_clusters * num_test_clusters, 0.f);

    std::vector<double> ref_normalization(num_ref_clusters, 0.f);
    std::vector<double> test_normalization(num_test_clusters, 0.f);


    for (int i = 0; i < NCaloCells; ++i)
      {
        const tag_type ref_tag = ref_tag_array[i];
        const tag_type test_tag = test_tag_array[i];
        const double SNR = std::abs(SNR_array[i]);
        const double weight = SNR * ( SNR > SNR_thresholds[2] ? seed_weight :
                                      (
                                      SNR > SNR_thresholds[1] ? grow_weight :
                                      (
                                      SNR > SNR_thresholds[0] ? term_weight :
                                      0
                                      )
                                      )
                                    );
        const int ref_idx = Tags::get_index_from_tag(ref_tag);
        const int test_idx = Tags::get_index_from_tag(test_tag);
        if (Tags::is_part_of_cluster(ref_tag) && Tags::is_part_of_cluster(test_tag))
          {
            similarity_map[test_idx * num_ref_clusters + ref_idx] += weight;
          }
        if (Tags::is_part_of_cluster(ref_tag))
          {
            ref_normalization[ref_idx] += weight;
          }
        if (Tags::is_part_of_cluster(test_tag))
          {
            test_normalization[test_idx] += weight;
          }
      }

    //In essence, the Gale-Shapley Algorithm

    std::vector<std::vector<int>> sorted_GPU_matches;

    sorted_GPU_matches.reserve(num_test_clusters);

    for (int testc = 0; testc < num_test_clusters; ++testc)
      {
        std::vector<int> sorter(num_ref_clusters);
        std::iota(sorter.begin(), sorter.end(), 0);

        std::sort(sorter.begin(), sorter.end(),
                  [&](const int a, const int b)
        {
          const double a_weight = similarity_map[testc * num_ref_clusters + a];
          const double b_weight = similarity_map[testc * num_ref_clusters + b];
          return a_weight > b_weight;
        }
                 );

        int wanted_size = 0;

        for (; wanted_size < sorter.size(); ++wanted_size)
          {
            const double match_weight = similarity_map[testc * num_ref_clusters + sorter[wanted_size]] / test_normalization [testc];
            if (match_weight < min_similarity)
              {
                break;
              }
          }

        //Yeah, we could do a binary search for best worst-case complexity,
        //but we are expecting 1~2 similar clusters and the rest garbage,
        //so we're expecting only 1~2 iterations.
        //This actually means all that sorting is way way overkill,
        //but we must make sure in the most general case that this works...

        sorter.resize(wanted_size);

        sorted_GPU_matches.push_back(sorter);
      }

    int num_iter = 0;

    constexpr int max_iter = 32;


    r2t_matches.clear();
    t2r_matches.clear();

    r2t_matches.resize(num_ref_clusters, -1);
    t2r_matches.resize(num_test_clusters, -1);


    std::vector<double> matched_weights(num_ref_clusters, -1.);

    std::vector<int> skipped_matching(num_test_clusters, 0);

    for (int stop_counter = 0; stop_counter < num_test_clusters && num_iter < max_iter; ++num_iter)
      {
        stop_counter = 0;
        for (int testc = 0; testc < sorted_GPU_matches.size(); ++testc)
          {
            if (skipped_matching[testc] < sorted_GPU_matches[testc].size())
              {
                const int match_c = sorted_GPU_matches[testc][skipped_matching[testc]];
                const double match_weight = similarity_map[testc * num_ref_clusters + match_c] / ref_normalization[match_c];
                if (match_weight >= min_similarity && match_weight > matched_weights[match_c])
                  {
                    const int prev_match = r2t_matches[match_c];
                    if (prev_match >= 0)
                      {
                        ++skipped_matching[prev_match];
                        --stop_counter;
                      }
                    r2t_matches[match_c] = testc;
                    matched_weights[match_c] = match_weight;
                    ++stop_counter;
                  }
                else
                  {
                    ++skipped_matching[testc];
                  }
              }
            else
              {
                ++stop_counter;
              }
          }
      }

    for (int i = 0; i < r2t_matches.size(); ++i)
      {
        const int match = r2t_matches[i];
        if (match >= 0)
          {
            t2r_matches[match] = i;
          }
      }

  }


  void add_comparison(const EventDataHolder & event_data, const ConstantDataHolder & instance_data)
  {

    hipDeviceSynchronize();

    Helpers::CPU_object<CellStateArr> cell_state = event_data.m_cell_state_dev;

    Helpers::CPU_object<ClusterInfoArr> cluster_info = event_data.m_clusters_dev;

    std::unordered_map<int, int> tag_map;

    {

      std::vector<int> cluster_order(cluster_info->number);

      std::iota(cluster_order.begin(), cluster_order.end(), 0);

      std::sort(cluster_order.begin(), cluster_order.end(), [&](const int a, const int b)
      {
        if (cluster_info->seedCellID[a] < 0)
          {
            return false;
            //This means that clusters with no cells
            //(marked as invalid) always compare lower,
            //so they appear in the end.
          }
        else if (cluster_info->seedCellID[b] < 0)
          {
            return true;
          }
        return cluster_info->clusterEt[a] > cluster_info->clusterEt[b];
      } );

      int real_cluster_numbers = cluster_info->number;

      for (size_t i = 0; i < cluster_order.size(); ++i)
        {
          const int this_id = cluster_order[i];
          if (cluster_info->seedCellID[this_id] < 0)
            {
              tag_map[this_id] = -1;
              --real_cluster_numbers;
            }
          else
            {
              tag_map[this_id] = i;
            }
        }

      const Helpers::CPU_object<ClusterInfoArr> temp_clusters(cluster_info);

      cluster_info->number = real_cluster_numbers;

      for (int i = 0; i < temp_clusters->number; ++i)
        {
          cluster_info->clusterEnergy[i] = temp_clusters->clusterEnergy[cluster_order[i]];
          cluster_info->clusterEt[i] = temp_clusters->clusterEt[cluster_order[i]];
          cluster_info->clusterEta[i] = temp_clusters->clusterEta[cluster_order[i]];
          cluster_info->clusterPhi[i] = temp_clusters->clusterPhi[cluster_order[i]];
          cluster_info->seedCellID[i] = temp_clusters->seedCellID[cluster_order[i]];
        }

    }
    for (int i = 0; i < NCaloCells; ++i)
      {
        const tag_type this_tag = cell_state->clusterTag[i];
        if (!Tags::is_part_of_cluster(this_tag))
          {
            cell_state->clusterTag[i] = Tags::InvalidTag;
          }
        else
          {
            const int old_idx = Tags::get_index_from_tag(this_tag);
            const int new_idx = tag_map[old_idx];
            if (new_idx < 0)
              {
                cell_state->clusterTag[i] = Tags::InvalidTag;
              }
            else
              {
                cell_state->clusterTag[i] = Tags::make_seed_tag(0x7f7fffff, cluster_info->seedCellID[new_idx], new_idx);
                //To match what we do on the CPU side...
              }
          }
      }

    std::vector<int> r2t_matches, t2r_matches;

    find_matches(r2t_matches, t2r_matches, m_snr_arr->snr, m_state->clusterTag, cell_state->clusterTag, m_ref_clusters, cluster_info->number);
    /*
    for (int i = 0; i < r2t_matches.size(); ++i)
      {
    std::cout << i << " " << r2t_matches[i] << "\n";
      }
    for (int i = 0; i < t2r_matches.size(); ++i)
      {
    std::cout << i << " " << t2r_matches[i] << "\n";
      }
    std::cout << std::endl;
    */

    std::vector<bool> equal_cells(t2r_matches.size(), true);

    size_t diffs = 0, seed_count = 0, grow_count = 0, term_count = 0;

    for (int i = 0; i < NCaloCells; ++i)
      {
        const tag_type test_real_tag = cell_state->clusterTag[i];
        const tag_type ref_real_tag = m_state->clusterTag[i];

        const int test_tag = ( Tags::is_part_of_cluster(test_real_tag) ?
                               Tags::get_index_from_tag(test_real_tag) : -1 );
        const int ref_tag = ( Tags::is_part_of_cluster(ref_real_tag) ?
                              Tags::get_index_from_tag(ref_real_tag) : -1 );

        if ((test_tag >= 0 && t2r_matches[test_tag] != ref_tag) || (test_tag < 0 && ref_tag >= 0))
          {
            equal_cells[ref_tag] = false;
            ++diffs;
            has_total = true;
            const float snr = m_snr_arr->snr[i];
            if (snr > 4)
              {
                has_seed = true;
                ++seed_count;
                //std::cout << i << ": " << test_tag << " " << ref_tag << " (" << snr << ")" << std::endl;
              }
            else if (snr > 2)
              {
                has_grow = true;
                //std::cout << i << ": " << test_tag << " " << ref_tag << " (" << snr << ")" << std::endl;
                ++grow_count;
              }
            else if (snr > 0)
              {
                has_term = true;
                ++term_count;
              }
            else
              {
                std::cout << "Hmmm " << i << std::endl;
              }
          }
      }

    total.push_back(diffs);
    seed.push_back(seed_count);
    grow.push_back(grow_count);
    term.push_back(term_count);

    std::vector< std::vector< std::vector<double> > > phi_calcs;

    {

      standard_phi_calc pc_s{};
      standard_phi_calc_no_reg pc_snr{};
      KBN_phi_calc pc_kbn{};
      KBN_phi_calc_no_reg pc_kbnnr{};
      sincos_phi_calc pc_sc{};
      sincos_KBN_phi_calc pc_sckbn{};

      std::vector<base_phi_calc *> calcs{& pc_s, & pc_snr, & pc_kbn, & pc_kbnnr, & pc_sc, & pc_sckbn};

      std::vector<const CellStateArr *> tag_arrays{m_state, cell_state};
      std::vector<int> num_state_clusters{m_ref_clusters, cluster_info->number};

      for (size_t i = 0; i < tag_arrays.size(); ++i)
        {
          phi_calcs.emplace_back();
          for (auto & calc_method : calcs)
            {
              phi_calcs.back().emplace_back();
              calculate_phis(*calc_method, phi_calcs.back().back(), *tag_arrays[i], num_state_clusters[i],
                             instance_data.m_geometry->phi, m_snr_arr->snr, event_data.m_cell_info->energy);
            }
        }

    }

    std::vector<phi_errors> phi_errs(phi_comparison_labels.size());

    constexpr double max_diff = 0.25;

    int count = 0;
    for (int test_id = 0; test_id < cluster_info->number; ++test_id)
      {
        if (t2r_matches[test_id] < 0)
          {
            continue;
          }
        const double GPU_phi = cluster_info->clusterPhi[test_id];

        std::vector<int> indices{t2r_matches[test_id], test_id};

        auto check = [&](const double val, const int id1, const int id2)
        {
          if (val < -Helpers::Constants::pi<double> || val >  Helpers::Constants::pi<double>)
            {
              std::cout << id1 << " " << id2 << ": " << val << std::endl;
              return true;
            }
          return false;
        };

        bool stop = false;

        if (check(GPU_phi, -1, -1))
          {
            stop = true;
          }


        for (int i = 0; i < phi_calcs.size(); ++i)
          {
            for (int j = 0; j < phi_calcs[i].size(); ++j)
              {
                if (check(phi_calcs[i][j][indices[i]], i, j))
                  {
                    stop = true;
                  }
              }
          }

        if (stop)
          {
            continue;
          }

        struct calc_spec
        {
          int origin;
          int calc;
        };

        struct diff_spec
        {
          calc_spec a, b;
        };

        std::vector<diff_spec> diffs{ {{0, 0}, {1, 0}},   //Reference to CPU-Calculated
          {{0, 0}, {-1, -1}}, //Reference to GPU-Calculated
          {{1, 0}, {-1, -1}}, //CPU-Calculated to GPU-Calculated
          {{1, 0}, {1, 1}},   //CPU to CPU No Reg.
          {{1, 0}, {1, 2}},   //CPU to CPU KBN
          {{1, 0}, {1, 3}},   //CPU to CPU KBN No Reg.
          {{1, 0}, {1, 4}},   //CPU to CPU Sin/Cos
          {{1, 0}, {1, 5}},   //CPU to CPU Sin/Cos KBN
          {{1, 3}, {-1, -1}}, //CPU KBN No Reg. to GPU
          {{0, 1}, {-1, -1}}, //Reference No Reg. to GPU
          {{0, 4}, {-1, -1}}  //Reference Sin/Cos to GPU
        };

        auto getty = [&](const calc_spec & cs)
        {
          if (cs.origin < 0 || cs.calc < 0)
            {
              return GPU_phi;
            }
          else
            {
              return phi_calcs[cs.origin][cs.calc][indices[cs.origin]];
            }
        };

        auto setty = [&](phi_errors & pe, const double delta)
        {
          pe.add(delta);
          if (delta > max_diff)
            {
              pe.add_difference();
            }
        };

        for (int i = 0; i < diffs.size(); ++i)
          {
            const double one = getty(diffs[i].a);
            const double two = getty(diffs[i].b);

            const double dif = Helpers::Constants::pi<double> - std::abs(std::fmod(std::abs(one - two), 2 * Helpers::Constants::pi<double>) - Helpers::Constants::pi<double>);

            setty(phi_errs[i], dif);
          }
        ++count;
      }

    valid_phis_counts.push_back(count);

    for (int i = 0; i < phi_errs.size(); ++i)
      {
        phi_errs[i].finalize(count);
        phi_comparisons[i].push_back(phi_errs[i]);
      }


  }


  void print_phi(const std::vector<phi_errors> & vect, const std::vector<size_t> & counts, const std::string & title, const size_t break_after)
  {
    std::cout << "\n\n " << title << ":";

    phi_errors accum{0, 0, 0}, local_accum{0, 0, 0};
    size_t counter = 0, local_counter = 0;

    for (size_t i = 0; i < vect.size(); ++i)
      {
        if (i % break_after == 0)
          {
            if (i > 0)
              {
                local_accum.finalize(local_counter, break_after);
                std::cout << "("  << local_accum << ")";
              }
            local_accum = phi_errors {0, 0, 0};
            local_counter = 0;
            std::cout << "\n             ";
          }
        const phi_errors res = vect[i];
        const size_t num = counts[i];
        printf("|%7.3f %7.3f %7.3f %7.3f| ", res.differences, res.max, res.mu + (res.mu < 1e-3) * 1e-3, std::sqrt(res.sigma_sqr));
        local_accum.combine(res, num);
        local_counter += num;
        accum.combine(res, num);
        counter += num;
      }
    local_accum.finalize(local_counter, break_after);
    std::cout << "(" << local_accum << ")";
    accum.finalize(counter, vect.size());
    std::cout << "\n (" << accum << ")";
  }


  void print_comparison(const size_t break_after, const bool force_print)
  {
    std::cout << " --------- CPU COMPARISON --------- \n\n";

    if (has_total || force_print)
      {
        generic_comparison_print(total, "Total", break_after);
      }

    if (has_seed || force_print)
      {
        generic_comparison_print(seed, "Seed", break_after);
      }


    if (has_grow || force_print)
      {
        generic_comparison_print(grow, "Grow", break_after);
      }

    if (has_term || force_print)
      {
        generic_comparison_print(term, "Terminal", break_after);
      }

    if (!has_total)
      {
        std::cout << "\nAll good, you can go rest now.\n";
      }

    std::cout << "\n\n --------- PHI COMPARISON --------- \n\n";

    for (size_t i = 0; i < phi_comparison_labels.size(); ++i)
      {
        print_phi(phi_comparisons[i], valid_phis_counts, phi_comparison_labels[i], break_after);
      }

    std::cout << std::endl;

  }

};

void setup_cuda_device()
{
  int devID = 0;
  hipDeviceProp_t props;

  /* maybe we want something else here */
  hipSetDevice(0);

  hipGetDeviceProperties(&props, devID);
  std::cout << "[CUDA] Device " << devID << " " << props.name <<  " with compute capability " << props.major << "." << props.minor << std::endl;
}

int main(int argc, char ** argv)
{
  if (argc < 3)
    {
      std::cout << "Expected arguments: <program> <num reps> <max events> <geometry and noise folder> <events folder>" << std::endl;
      return 0;
    }

  setup_cuda_device();

  const size_t num_reps = std::strtoull(argv[1], nullptr, 10);

  const int max_events = std::atoi(argv[2]);
  
  const auto constants_folder = StandaloneDataIO::load_folder(argv[3], 0, false, false, true, true);

  const auto loaded_folder = StandaloneDataIO::load_folder(argv[4], max_events, true, true, false, false);

  ConstantDataHolder fixed_data;

  fixed_data.m_geometry = constants_folder.geometry.begin()->second;
  fixed_data.m_cell_noise = constants_folder.noise.begin()->second;

  fixed_data.sendToGPU(false);

  TACTemporariesHolder temporary_holder;

  temporary_holder.allocate();

  TACOptionsHolder options_holder;

  options_holder.allocate();

  *(options_holder.m_options) = TopoAutomatonOptions { 4.0f, 2.0f, 0.0f, true, true, true, false, 0x7FFFFFFF};

  options_holder.sendToGPU();

  BasicGPUClusterInfoCalculatorTemporariesHolder other_temporary_holder;
  
  other_temporary_holder.allocate();


  Helpers::CPU_object<CellStateArr> temp_cells;
  Helpers::CPU_object<TopoAutomatonTemporaries> temp_temp;

  temp_cells.allocate();
  temp_temp.allocate();

  EventDataHolder event_data;
  event_data.allocate();

  std::vector<size_t> times;

  Results results;
  CPUComparison comparison;


  for (const auto & it : loaded_folder.cell_info)
    {
      event_data.m_cell_info = (*it.second);

      event_data.sendToGPU();

      //CUDA_ERRCHECK(hipDeviceSynchronize());

      //std::cout << "Sent data." << std::endl;

      signalToNoise(event_data, temporary_holder, fixed_data, options_holder, true);

      //CUDA_ERRCHECK(hipDeviceSynchronize());

      cellPairs(event_data, temporary_holder, fixed_data, options_holder, true);

      //CUDA_ERRCHECK(hipDeviceSynchronize());

      temp_cells = event_data.m_cell_state_dev;
      temp_temp = temporary_holder.m_temporaries_dev;
      

      for (size_t rep = 0; rep <= num_reps; ++rep)
        {

          //CUDA_ERRCHECK(hipDeviceSynchronize());

          if (rep != 0)
            {
              event_data.m_cell_state_dev = temp_cells;
              temporary_holder.m_temporaries_dev = temp_temp;
            }

          //CUDA_ERRCHECK(hipDeviceSynchronize());


          //std::cout << "So far, so good... " << rep << std::endl;

          auto start = std::chrono::steady_clock::now();
          
          clusterGrowing(event_data, temporary_holder, fixed_data, options_holder, true);
          
          auto end = std::chrono::steady_clock::now();
          
          times.push_back(std::chrono::duration_cast<std::chrono::microseconds>(end - start).count());

          //CUDA_ERRCHECK(hipDeviceSynchronize());
          if (rep == 0)
            {
              updateSeedCellProperties(event_data, other_temporary_holder, fixed_data, true);
              calculateClusterProperties(event_data, other_temporary_holder, fixed_data, true);
              
              results.set_reference(event_data);
              comparison.set_reference(
              *(loaded_folder.cell_state.at(it.first)),
              (*it.second),
              fixed_data,
              loaded_folder.clusters.at(it.first)->number
              );
              comparison.add_comparison(event_data, fixed_data);
            }
          else
            {
              results.add_comparison(event_data);
            }
        }
    }
    
  generic_comparison_print(times, "Execution Times", num_reps);
  std::cout << "\n";
  results.print_comparison(num_reps, false);
  comparison.print_comparison(loaded_folder.cell_info.size(), true);

  //CUDA_ERRCHECK(hipDeviceSynchronize());

  return 0;
}