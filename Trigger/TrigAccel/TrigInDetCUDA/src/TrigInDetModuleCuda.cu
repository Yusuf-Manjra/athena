/*
  Copyright (C) 2002-2020 CERN for the benefit of the ATLAS collaboration
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <atomic>

#include "TrigInDetModuleCuda.h"
#include "SeedMakingDataStructures.h"
#include "SeedMakingWorkCuda.h"

#include "TrigAccelEvent/TrigInDetAccelCodes.h"

extern "C" TrigAccel::WorkFactory* getFactory() {
  return new TrigInDetModuleCuda();
}

extern "C" int getFactoryId() { 
  return TrigAccel::TrigInDetModuleID_CUDA; 
} 	  

extern "C" void deleteFactory(TrigAccel::WorkFactory* c){
  TrigInDetModuleCuda* mod=reinterpret_cast<TrigInDetModuleCuda*>(c);
  delete mod;
}

TrigInDetModuleCuda::TrigInDetModuleCuda() : m_maxNumberOfContexts(12), m_maxDevice(0), m_usePinnedMemory(true), 
						  m_useWriteCombinedMemory(false),  m_linkOutputToShm(false), m_dumpTimeLine(false) {

  m_d_detmodels.clear();

  hipGetDeviceCount(&m_maxDevice);

  hipError_t error = hipGetLastError();

  if(error != hipSuccess) {
    m_maxDevice = 0;
  }

  for(unsigned int i=0;i<getProvidedAlgs().size();i++) {
    m_workItemCounters[i] = 0;
  }

  m_timeLine.clear();

}

TrigInDetModuleCuda::~TrigInDetModuleCuda() {

  SeedMakingDeviceContext* ps = 0;
  
  std::cout<<"deleting "<<m_seedMakingDcQueue.unsafe_size()<<" device contexts"<<std::endl;
  
  while(m_seedMakingDcQueue.try_pop(ps)) deleteSeedMakingContext(ps);

  for(auto dm : m_d_detmodels) {
    hipSetDevice(dm.first);
    hipFree(dm.second);

  }
  m_d_detmodels.clear();

  if(m_dumpTimeLine) {

     std::cout<<"time_line has "<<m_timeLine.size()<<" events"<<std::endl;
     if(m_timeLine.size() > 0) {
        tbb::tick_count t0 = m_timeLine[0].m_time;
        std::ofstream tl("timeLine.csv");
        tl<<"workId,eventType,time"<<std::endl;
        tl<<m_timeLine[0].m_workId<<","<<m_timeLine[0].m_eventType<<",0"<<std::endl;
        for(unsigned int tIdx = 1;tIdx < m_timeLine.size();++tIdx) {
           tbb::tick_count t1 = m_timeLine[tIdx].m_time;
           auto duration = t1-t0;
           tl<<m_timeLine[tIdx].m_workId<<","<<m_timeLine[tIdx].m_eventType<<","<<1000*duration.seconds()<<std::endl;
        }
        tl.close();
     }
  }
}

bool TrigInDetModuleCuda::configure() {

  std::vector<int> allowedGPUs, nDeviceContexts;

  allowedGPUs.resize(1,0);//configured for just 1 device with deviceId = 0

  nDeviceContexts.resize(1,8);//configured for 8 DataContexts

  if(m_maxDevice == 0) {
     std::cout<<"No CUDA devices found"<<std::endl;
     return false;
  }
  
  if(allowedGPUs.empty() || nDeviceContexts.empty()) return false;

  if(allowedGPUs.size() != nDeviceContexts.size()) return false;

  unsigned int dcIndex=0;

  size_t memTotalSize = 0;

  std::vector< SeedMakingDeviceContext*> vSeedDCs[100];//we do not have that many GPUs

  int nDCTotal=0;

  for(std::vector<int>::iterator devIt = allowedGPUs.begin(); devIt!= allowedGPUs.end();++devIt, dcIndex++) {

    int deviceId = (*devIt);

    if(deviceId<0 || deviceId>=m_maxDevice) continue;

    size_t memTotalSizeOnDevice = 0;

    hipSetDevice(deviceId);

    checkError();
        
    unsigned char* d_detmodel;

    hipMalloc((void **)&d_detmodel, sizeof(TrigAccel::DETECTOR_MODEL));

    checkError();
    
    m_d_detmodels.insert(std::pair<unsigned int, unsigned char*>(deviceId, d_detmodel));
    
    int nDC = nDeviceContexts[dcIndex];
    nDCTotal += nDC;

    memTotalSizeOnDevice += sizeof(TrigAccel::DETECTOR_MODEL);

    for(int dc=0;dc<nDC;dc++) {
      SeedMakingDeviceContext* p = createSeedMakingContext(deviceId);
      memTotalSizeOnDevice += p->deviceSize();
      vSeedDCs[dcIndex].push_back(p);
    }

    memTotalSize += memTotalSizeOnDevice;
    
    std::cout<<"GPU"<<deviceId<<" allocated data context size = "<<1e-6*memTotalSizeOnDevice<<" MBytes"<<std::endl;
  }

  int nDCLeft = nDCTotal;
  while(nDCLeft>0) {
     for(unsigned int iGPU=0;iGPU<allowedGPUs.size();iGPU++) {
        if(vSeedDCs[iGPU].empty()) continue;
	m_seedMakingDcQueue.push(vSeedDCs[iGPU].back());
	vSeedDCs[iGPU].pop_back();
        --nDCLeft;
     }
  }
  
  std::cout<<"Data context queue : ";
  for(tbb::concurrent_queue< SeedMakingDeviceContext*>::const_iterator i(m_seedMakingDcQueue.unsafe_begin()); i!=m_seedMakingDcQueue.unsafe_end(); ++i ) {
    std::cout<<(*i)->m_deviceId<<" ";
  }
  std::cout<<std::endl;

  std::cout<<"Total size of memory allocated on all GPUs = "<<1e-6*memTotalSize<<" MBytes"<<std::endl;

  return true;
}


SeedMakingDeviceContext* TrigInDetModuleCuda::createSeedMakingContext(int id) {

  hipSetDevice(id);

  SeedMakingDeviceContext* p = new SeedMakingDeviceContext;

  p->m_deviceId = id;

  //set stream

  hipStreamCreate(&p->m_stream);

  //check device property

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, id);

  p->m_gpuParams.m_nSMX = deviceProp.multiProcessorCount;

  int ncores = 0;

  if ((deviceProp.minor == 1) || (deviceProp.minor == 2)) ncores = 128;
  else if (deviceProp.minor == 0) ncores = 64;	   
       else printf("Cannot determine the number of cores: unknown device type\n"); 

  p->m_gpuParams.m_nNUM_SMX_CORES = ncores;//_ConvertSMVer2Cores_local(deviceProp.major, deviceProp.minor);
  p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = NUM_TRIPLET_BLOCKS;
  if(deviceProp.maxThreadsPerBlock < p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS) 
    p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = deviceProp.maxThreadsPerBlock;

  //Allocate memory
  
  hipMalloc((void **)&p->d_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipMalloc((void **)&p->d_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));

  auto dmIt = m_d_detmodels.find(p->m_deviceId);
  if(dmIt!=m_d_detmodels.end()) {
    p->d_detmodel = (*dmIt).second;
  }
  
  hipMalloc((void **)&p->d_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));
  hipMalloc((void **)&p->d_doubletstorage, sizeof(DOUBLET_STORAGE));
  hipMalloc((void **)&p->d_doubletinfo, sizeof(DOUBLET_INFO));


  p->d_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) +  
              sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE) + sizeof(DOUBLET_STORAGE) + sizeof(DOUBLET_INFO);

  hipHostMalloc((void **)&p->h_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipHostMalloc((void **)&p->h_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));
  hipHostMalloc((void **)&p->h_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));

  p->h_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) + sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE);

  return p;
}

void TrigInDetModuleCuda::deleteSeedMakingContext(SeedMakingDeviceContext* p) {

  int id = p->m_deviceId;

  hipSetDevice(id);

  hipStreamDestroy(p->m_stream);

  hipFree(p->d_settings);
  hipFree(p->d_spacepoints);
  
  hipFree(p->d_outputseeds);
  hipFree(p->d_doubletstorage);
  hipFree(p->d_doubletinfo);

  hipHostFree(p->h_settings);
  hipHostFree(p->h_spacepoints);
  hipHostFree(p->h_outputseeds);

  delete p;

}


TrigAccel::Work* TrigInDetModuleCuda::createWork(int workType, std::shared_ptr<TrigAccel::OffloadBuffer> data){
  
  if(workType == TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT){
    
    for(auto dm : m_d_detmodels) {

       unsigned int deviceId = dm.first;

       hipSetDevice(deviceId);
       
       hipMemcpy(dm.second, (unsigned char*)data->get(), sizeof(TrigAccel::DETECTOR_MODEL), hipMemcpyHostToDevice);
    }
    return 0;
  }

  if(workType == TrigAccel::InDetJobControlCode::MAKE_SEEDS){
 
    SeedMakingDeviceContext* ctx = 0;
    
    while(!m_seedMakingDcQueue.try_pop(ctx)) {
      //      std::cout<<"waiting for free device context..."<<std::endl;
    };

    TrigAccel::SEED_MAKING_JOB *pArray = reinterpret_cast<TrigAccel::SEED_MAKING_JOB*>(data->get());
    
    //1. copy settings to the context host array

    TrigAccel::SEED_FINDER_SETTINGS* p_settings = reinterpret_cast<TrigAccel::SEED_FINDER_SETTINGS*>(ctx->h_settings);
    memcpy(p_settings, &pArray->m_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));

    //2. copy spacepoints to the context host array

    TrigAccel::SPACEPOINT_STORAGE* p_spacePoints = reinterpret_cast<TrigAccel::SPACEPOINT_STORAGE*>(ctx->h_spacepoints);
    memcpy(p_spacePoints, &pArray->m_data, sizeof(TrigAccel::SPACEPOINT_STORAGE));
    
    unsigned int workNum = m_workItemCounters[0]++;//seed making uses counter #0
    
    unsigned int workId = workNum*100;
    
    SeedMakingWorkCuda* w = new SeedMakingWorkCuda(workId, SeedMakingWorkContextCuda(ctx, m_usePinnedMemory, 
									     m_useWriteCombinedMemory, 
									     m_linkOutputToShm), data, m_seedMakingDcQueue, m_timeLine);
    
    return w;
  }

  return 0;
}

const std::vector<int> TrigInDetModuleCuda::getProvidedAlgs(){
  std::vector<int> v{
      TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT,
      TrigAccel::InDetJobControlCode::MAKE_SEEDS
  };
  return v;
}

